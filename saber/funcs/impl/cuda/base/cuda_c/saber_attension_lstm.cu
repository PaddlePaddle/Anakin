#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_attension_lstm.h"
#include "saber/core/tensor_op.h"
#include "saber/funcs/debug.h"
#include "hip/hip_fp16.h"

namespace anakin {

namespace saber {
static void gemm(hipblasHandle_t handle, 
                 const bool TransA, const bool TransB, 
                 int m, int n, int k, const float alpha,
                 const float* a, const float* b, 
                 const float beta, float* c) {
    //    cout << "(" << m << "," << n << "," << k << ")" << endl;
    int lda = (!TransA/* == CblasNoTrans*/) ? k : m;
    int ldb = (!TransB/* == CblasNoTrans*/) ? n : k;
    hipblasOperation_t cuTransA =
        (!TransA/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        (!TransB/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasSgemm(handle, cuTransA, cuTransB, m, n, k, &alpha, b, ldb, a, lda, &beta, c, n);
};
/*one block compute one sequence*/
/*use share memory to reduce*/
template <typename Dtype>
__global__ void sequence_softmax(const Dtype* in_data, const int* seq_offset, const int seq_num, Dtype* out_data) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (t_id >= seq_num) {
        return;
    }
    int start_id = seq_offset[t_id];
    int end_id = seq_offset[t_id + 1];
     Dtype max_data = -1e32;
    //Dtype max_data = -FLT_MAX;
    Dtype sum = 0;
    for (int i = start_id; i < end_id; i++) {
        max_data = in_data[i] > max_data ? in_data[i] : max_data;
    }
    for (int i = start_id; i < end_id; i++) {
        sum +=  expf(in_data[i] - max_data);
    }
    for (int i = start_id; i < end_id; i++) {
        out_data[i] =  expf(in_data[i] - max_data) / sum;
    }
}
template <typename Dtype>
__global__ void relu(const Dtype* in_data, Dtype* out_data, int count) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (t_id >= count) {
        return;
    }
    out_data[t_id] = in_data[t_id] > 0 ? in_data[t_id] : 0;
}

template <typename Dtype>
__global__ void bias_relu(const Dtype* in_data, const Dtype* bias_data, const int count, const int bias_size, Dtype* out_data) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (t_id >= count) {
        return;
    }
    int id = t_id % bias_size;
    Dtype data = in_data[t_id] + bias_data[id]; 
    out_data[t_id] = data > 0 ? data : 0;
}

template <typename Dtype>
__global__ void sequence_pool(const Dtype* in_data, const Dtype* scale, const int* seq_offset, const int seq_num, const int total_num, const int dim, Dtype* out_data) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (t_id >= seq_num * dim) {
        return;
    }
    int dim_id = t_id % dim;
    int seq_id = t_id / dim;
    int start_id = seq_offset[seq_id];
    int end_id = seq_offset[seq_id + 1];
    Dtype sum = 0;
    const Dtype * in = in_data + dim_id + start_id * dim;
    for (int i = 0; i < end_id - start_id; i++) {
        sum += in[0] * scale[i + start_id];
        in += dim;
    }
    out_data[t_id] = sum;
}
template<typename Dtype>
__device__ Dtype sigmoid(Dtype in) {
    Dtype out = Dtype(1.0) / (1 + exp(-in));
    return out;
}

template<typename Dtype>
__device__ Dtype tanh(Dtype in) {
    //Dtype out = (exp(in)- exp(-in)) / (exp(in) + exp(-in));
    //Dtype out = 1 - 2.f / (expf(2*in) + 1);
    //Dtype out = 1 - 2.f / (expf(2*in) + 1);
    Dtype a = expf(in);
    Dtype b = expf(-in);
    return  (a- b)/(a+b);
}

template <typename Dtype>
__global__ void lstm_bias_and_act(const Dtype* in_data, const Dtype* bias_data, Dtype* out_data, Dtype* cell_data, int batch_size, int hidden_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= batch_size * hidden_size) {
        return;
    }
    int dim_id = tid % hidden_size;
    int batch_id  = tid / hidden_size;
    int offset = batch_id * hidden_size + dim_id;
    const Dtype* tmp_in = in_data + batch_id * 4 * hidden_size + dim_id;
    Dtype* tmp_cell = cell_data + offset;
    const Dtype* tmp_bias = bias_data + dim_id;
    Dtype ct  = tanh(tmp_in[2 * hidden_size] + tmp_bias[2 * hidden_size]);
    Dtype ig  = sigmoid(tmp_in[0 * hidden_size] + tmp_bias[0 * hidden_size]);
    Dtype fg  = sigmoid(tmp_in[1 * hidden_size] + tmp_bias[1 * hidden_size]);
    Dtype og  = sigmoid(tmp_in[3 * hidden_size] + tmp_bias[3 * hidden_size]);
    tmp_cell[0] = ig * ct  + fg * tmp_cell[0];
    out_data[offset] = og * tanh(tmp_cell[0]);
}

template <typename Dtype>
__global__ void sequence_bias_relu(const Dtype* in_data, const Dtype* seq_bias,  const Dtype* bias_data, const int * seq_id, const int num, const int dim,
                 Dtype* out_data) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= dim * num) {
        return;
    }
    int dim_id = tid % dim;
    int word_id = tid / dim;
    int cur_seq_id = seq_id[word_id];
    Dtype data  = in_data[tid] + seq_bias[cur_seq_id * dim + dim_id] + bias_data[dim_id];
    //printf("%d, in:%f,  seq_bias:%f, bias:%f\n", tid, in_data[tid], seq_bias[cur_seq_id * dim + dim_id], bias_data[dim_id]);
    out_data[tid] = data > 0 ? data : 0;
}

template <typename Dtype>
__global__ void lstm_result_to_sequence(const Dtype * in_data, const int* seq_id_map, const int* offset, const int seq_num,
                const int word_num, const int hidden_size, Dtype* out_data) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= hidden_size * word_num) {
        return;
    }
    int dim_id = tid % hidden_size;
    int word_id = tid / hidden_size;
    int seq_id = seq_id_map[word_id];
    int word_id_in_seq = word_id - offset[seq_id];
    out_data[tid] = in_data[(word_id_in_seq * seq_num  + seq_id) * hidden_size + dim_id];
}

template<>
    SaberStatus SaberAttensionLstm<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
                  const std::vector<DataTensor_in*>& inputs,
                  std::vector<DataTensor_out*>& outputs,
                  AttensionLstmParam <OpTensor>& param) {
    hipStream_t stream = this->_ctx->get_compute_stream();
    auto attn_param = param.attension_param;
    auto lstm_param = param.lstm_param;
    int hidden_size = lstm_param.with_peephole ? lstm_param.bias()->valid_size() / 7 : lstm_param.bias()->valid_size() / 4;

    DataTensor_in* input = inputs[0];
    _attn_outs.resize(attn_param.fc_vec.size());
    int seq_num = inputs[0]->get_seq_offset().size() - 1;
    int word_num = inputs[0]->num();
    Shape softmax_out_shape = {word_num, 1, 1, 1};
    _softmax_out.reshape(softmax_out_shape);
    _dev_seq_id_map.reshape(softmax_out_shape);
    std::vector<int> id_map;
    int seq_id = 0;
    auto seq_offset = inputs[0]->get_seq_offset();
    int max_len = 0;

    for (int i = 0; i < seq_num; i++) {
        for (int j = seq_offset[i]; j < seq_offset[i+1]; j++) {
            id_map.push_back(i);
        }
    }
    for (int i = 0; i < seq_num; i++) {
        int cur_len = seq_offset[i+ 1] - seq_offset[i];
        max_len = max_len < cur_len ? cur_len : max_len; 
    }
    hipMemcpyAsync(_dev_seq_id_map.mutable_data(), &id_map[0], sizeof(int) * word_num, hipMemcpyHostToDevice, stream);
    Shape offset_shape = {seq_num + 1, 1, 1, 1};
    _dev_offset.reshape(offset_shape);
    hipMemcpyAsync((void*)_dev_offset.mutable_data(), (void*)&seq_offset[0], 
            sizeof(int) * seq_offset.size(),
            hipMemcpyHostToDevice, stream); 
    
    /*for first fc*/
    
    int M_0 = input->num();
    int N_0 = attn_param.fc_vec[0].num_output;
    int K_0 = input->valid_size() / input->num();
    Shape first_fc_out_0_shape = {M_0, N_0, 1, 1};
    _first_fc_out_0.reshape(first_fc_out_0_shape);
    auto data_in = input->data();
    auto data_out = _first_fc_out_0.mutable_data();
    auto fc_vec = attn_param.fc_vec;

    //auto first_fc_0_kernel = saber_find_fast_sass_gemm(false, !fc_vec[0].is_transpose_weights, M_0, N_0, K_0);
    auto first_fc_0_kernel = saber_find_fast_sass_gemm(false, false, M_0, N_0, K_0);
    first_fc_0_kernel(M_0, N_0, K_0, 1.0f, data_in, 0.f, fc_vec[0].weights->data(), data_out, stream);
    Shape cell_shape = {seq_num, hidden_size, 1, 1};
    _cell_out.reshape(cell_shape);
    hipMemsetAsync(_cell_out.mutable_data(), 0, sizeof(float) * _cell_out.valid_size(), stream);
    Shape lstm_mid_shape = {seq_num, 4 * hidden_size, 1, 1};
    _hidden_out.reshape(lstm_mid_shape);
    Shape lstm_shape = {max_len * seq_num, hidden_size, 1, 1};
    
    _lstm_out.reshape(lstm_shape);
    

    /*for other fc*/
    for (int word_id = 0; word_id < max_len; word_id++) {
        _attn_outs[0]->reshape(first_fc_out_0_shape);
        
        if (word_id > 0) {
            Shape h_shape = {seq_num,  N_0, 1, 1};
            _first_fc_out_1.reshape(h_shape);
     
            auto kernel_1 = saber_find_fast_sass_gemm(false, false, seq_num, N_0, hidden_size);
            kernel_1(seq_num, N_0, hidden_size, 1.0f, 
                _cell_out.data(), 0.f, 
               fc_vec[0].weights->data() + K_0 * N_0,  _first_fc_out_1.mutable_data(), stream);

            sequence_bias_relu<<<CUDA_GET_BLOCKS(_attn_outs[0]->valid_size()), CUDA_NUM_THREADS, 0, stream>>>(_first_fc_out_0.data(), _first_fc_out_1.data(), fc_vec[0].bias->data(),
               _dev_seq_id_map.data(), M_0, N_0, _attn_outs[0]->mutable_data());
            
        } else {
            hipMemcpyAsync((void*)_attn_outs[0]->mutable_data(), (void*) _first_fc_out_0.data(), sizeof(float) * _attn_outs[0]->valid_size(), 
                hipMemcpyDeviceToDevice, stream);
            bias_relu<<<CUDA_GET_BLOCKS(_attn_outs[0]->valid_size()), CUDA_NUM_THREADS, 0, stream>>>(data_out, fc_vec[0].bias->data(), _attn_outs[0]->valid_size(), N_0, _attn_outs[0]->mutable_data());
        }
        
        for (int i = 1; i < attn_param.fc_vec.size(); i++) {
            int M = input->num();
            int N = attn_param.fc_vec[i].num_output;
            int K = attn_param.fc_vec[i-1].num_output;
            Shape attn_out_shape = {M, N, 1, 1};
            _attn_outs[i]->reshape(attn_out_shape);
            auto fc_in_data = _attn_outs[i - 1]->data();
            auto fc_out_data = _attn_outs[i]->mutable_data();

            auto kernel = saber_find_fast_sass_gemm(false, false, M, N, K);
            kernel(M, N, K, 1.0f, fc_in_data, 0.0f, fc_vec[i].weights->data(), fc_out_data, stream);
            bias_relu<<<CUDA_GET_BLOCKS(_attn_outs[i]->valid_size()), CUDA_NUM_THREADS, 0, stream>>>(fc_out_data, fc_vec[i].bias->data(), _attn_outs[i]->valid_size(), N, fc_out_data);
        }
        

        int fc_num = attn_param.fc_vec.size();
        int dim = inputs[0]->valid_size() / inputs[0]->num();

        Shape pool_shape = {seq_num, dim, 1, 1};
        _pool_out.reshape(pool_shape);
        
        sequence_softmax<<<CUDA_GET_BLOCKS(seq_num), CUDA_NUM_THREADS, 0, stream>>>(_attn_outs[fc_num - 1]->data(), _dev_offset.data(), seq_num, _softmax_out.mutable_data());

        sequence_pool<<<CUDA_GET_BLOCKS(seq_num * dim), CUDA_NUM_THREADS, 0, stream>>>(input->data(), _softmax_out.data(), _dev_offset.data(), seq_num, inputs[0]->num(), dim, _pool_out.mutable_data());
        
        
        auto  x_data = _pool_out.data();
        auto  _wx_data = lstm_param.weight()->data();
        auto  _bias_data = lstm_param.bias()->data();
        int word_size = dim;
        auto _wh_data = lstm_param.weight()->data() + 4 * hidden_size *  word_size;
        _gemm_wx(seq_num, 4 * hidden_size, word_size, 1.0, x_data, 0.0, lstm_param.weight()->data(), 
            _hidden_out.mutable_data(), stream);
        if (word_id > 0) {
            _gemm_wh(seq_num, 4 * hidden_size, hidden_size, 1.0, _lstm_out.data() + (word_id - 1) * seq_num * hidden_size, 1.0, _wh_data, _hidden_out.mutable_data(), stream);
        }
        auto cell_data = _cell_out.mutable_data();
        
        lstm_bias_and_act<<<CUDA_GET_BLOCKS(seq_num * hidden_size), CUDA_NUM_THREADS, 0, stream>>>(_hidden_out.data(), _bias_data, _lstm_out.mutable_data() + word_id * seq_num * hidden_size, cell_data, seq_num, hidden_size); 
    }
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipPeekAtLastError());
    lstm_result_to_sequence<<<CUDA_GET_BLOCKS(word_num * hidden_size), CUDA_NUM_THREADS, 0, stream>>>(_lstm_out.mutable_data(), _dev_seq_id_map.data(), 
        _dev_offset.data(), seq_num, word_num, hidden_size, outputs[0]->mutable_data());

    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}

}

}


