#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_scale.h"
#include "hip/hip_fp16.h"

namespace anakin{
namespace saber{

template<typename Dtype>
__global__ void ker_scale_fwd(Dtype * out_data,
                   const Dtype* in_data,
                   const Dtype* scale_data,
                   const Dtype* bias_data,
                   const int count,
                   const int scale_dim,
                   const int inner_dim) {
    CUDA_KERNEL_LOOP(tid, count){
        int scale_id = (tid / inner_dim) % scale_dim;
        Dtype scale = scale_data[scale_id];
        if (bias_data == nullptr) {
             out_data[tid] = scale * in_data[tid];
        } else {
             out_data[tid] = scale * in_data[tid] + bias_data[scale_id];
        }
    }
}

template<typename Dtype>
__global__ void ker_scale_fwd(Dtype * out_data,
                   const Dtype* in_data,
                   const Dtype scale,
                   const Dtype bias,
                   const int count) {
    CUDA_KERNEL_LOOP(tid, count){
        out_data[tid] = scale * in_data[tid] + bias;
    }
}


template <>
SaberStatus SaberScale<NV, AK_FLOAT>::dispatch( \
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ScaleParam<NV>& param) {

    hipStream_t hip_stream = this->_ctx->get_compute_stream();

    auto in_data = inputs[0]->data();
    auto out_data = outputs[0]->mutable_data();
    const int count = inputs[0]->valid_size();
    if (inputs.size() > 1) {
        _scale_dim = inputs[1]->valid_size();
        _inner_dim = count / _scale_dim;
    }
    if (_scale_dim > 1 || inputs.size() > 1) {
        auto scale_data = inputs.size() > 1 ? inputs[1]->data() : _weight.data();
        auto bias_data = param.bias_term ? _bias.data() : NULL;
        ker_scale_fwd<OpDataType>
                <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                (OpDataType*)out_data, (const OpDataType*)in_data, (const OpDataType*)scale_data, \
                (const OpDataType*)bias_data, count, _scale_dim, _inner_dim);
    } else {
        auto scale = param.scale_w[0];
        OpDataType bias = 0;
        if (_bias_term) {
            bias = param.scale_b[0];
        }
        ker_scale_fwd<OpDataType>
                <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                (OpDataType*)out_data, (const OpDataType*)in_data, scale, bias, count);
    }

    CUDA_POST_KERNEL_CHECK;
    return SaberSuccess;
}

}
}
