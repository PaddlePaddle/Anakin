#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_concat.h"

namespace anakin{

namespace saber{

const int BLOCK_SIZE = 32;

template <typename dtype>
__global__ void concat_impl_cuda(const int nthreads, const dtype* in_data,
                            const int num_concats, const int concat_size,
                            const int top_concat_axis, const int bottom_concat_axis,
                            const int offset_concat_axis, dtype* out_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int total_concat_size = concat_size * bottom_concat_axis;
        const int concat_num = index / total_concat_size;
        const int concat_index = index % total_concat_size;
        const int top_index = concat_index +
                              (concat_num * top_concat_axis + offset_concat_axis) * concat_size;

        out_data[top_index] = in_data[index];
    }
}

template <typename dtype>
__global__ void concat_impl_2d_impl(const int inner_size, const int num_concats,
                                    const dtype* in_data, const int concat_size,
                                    const int out_concat_axis,
                                    const int offset_concat_axis, dtype* out_data) {

    int idx_inner = threadIdx.x + blockIdx.x * blockDim.x;
    int idx_outer = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx_inner < inner_size && idx_outer < num_concats) {
        int idx_input = idx_outer * inner_size + idx_inner;
        int idx_output = (idx_outer * out_concat_axis + offset_concat_axis) * \
            concat_size + idx_inner;
        out_data[idx_output] = in_data[idx_input];
    }

}


template <>
SaberStatus SaberConcat<NV, AK_FLOAT>::dispatch(const std::vector<Tensor<NV> *>& inputs,
    std::vector<Tensor<NV> *>& outputs, ConcatParam<NV>& param) {

    hipStream_t stream = this->_ctx->get_compute_stream();

    int input_size = inputs.size();

    //! get output data, valid shape and stride shape
    OpDataType* out_data = (OpDataType*)outputs[0]->mutable_data();
    int offset_concat_axis = 0;
    Shape out_shape = outputs[0]->valid_shape();
    const int out_concat_axis = out_shape[param.axis];
    bool out_cont_flag = outputs[0]->is_continue_mem();
    bool in_cont_flag = inputs[0]->is_continue_mem();
    for (int i = 1; i < input_size; ++i) {
        in_cont_flag &= inputs[i]->is_continue_mem();
    }

    //! inputs and outputs are all with continuous memory
    if (in_cont_flag && out_cont_flag){
        for (int i = 0; i < input_size; ++i) {
            Shape in_shape = inputs[i]->valid_shape();
            //std::vector<int> bottom_shape = {tmp[3], tmp[2], tmp[1], tmp[0]};
            const OpDataType* in_data = (const OpDataType*)inputs[i]->data();
            const int in_concat_axis = in_shape[param.axis];
            const int in_concat_size = in_concat_axis * _concat_input_size;
            const int nthreads = in_concat_size * _num_concats;
            float ratio = (float)in_concat_size / _num_concats;
            bool is_balance = (ratio > 0.1 && ratio < 10);
            if (is_balance){
                int block_x = BLOCK_SIZE;
                int block_y = BLOCK_SIZE;
                int grid_x = (in_concat_size + block_x - 1) / block_x;
                int grid_y = (_num_concats + block_y - 1) / block_y;
                dim3 block(block_x, block_y);
                dim3 grid(grid_x, grid_y);
                concat_impl_2d_impl<OpDataType><<<grid, block, 0, stream>>>(
                        in_concat_size, _num_concats, in_data, _concat_input_size,
                                out_concat_axis, offset_concat_axis, out_data
                );
            } else {
                // NOLINT_NEXT_LINE(whitespace/operators)
                concat_impl_cuda<OpDataType><<<CUDA_GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>( \
                    nthreads, in_data, _num_concats, _concat_input_size, \
                            out_concat_axis, in_concat_axis, offset_concat_axis, out_data);
            }
            offset_concat_axis += in_concat_axis;
        }
    } else { //! inputs or outputs memory is not continuous
        Shape offset_out = outputs[0]->offset();
        Tensor<NV>  tsub;
        for (int i = 0; i < input_size; ++i) {
            Shape in_shape = inputs[i]->valid_shape();
            tsub.share_sub_buffer(*outputs[0], in_shape, offset_out);
            offset_out[param.axis] += in_shape[param.axis];
            tsub.async_copy_from(*inputs[i], stream);
        }
    }
    return SaberSuccess;
}
DEFINE_OP_TEMPLATE(SaberConcat, ConcatParam, NV, AK_INT8);
DEFINE_OP_TEMPLATE(SaberConcat, ConcatParam, NV, AK_HALF);
} //namespace anakin

} //namespace anakin
