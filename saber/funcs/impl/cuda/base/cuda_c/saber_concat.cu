#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_concat.h"

namespace anakin{

namespace saber{

const int BLOCK_SIZE = 32;

template <typename dtype>
__global__ void concat_impl_cuda(const int nthreads, const dtype* in_data,
                            const int num_concats, const int concat_size,
                            const int top_concat_axis, const int bottom_concat_axis,
                            const int offset_concat_axis, dtype* out_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int total_concat_size = concat_size * bottom_concat_axis;
        const int concat_num = index / total_concat_size;
        const int concat_index = index % total_concat_size;
        const int top_index = concat_index +
                              (concat_num * top_concat_axis + offset_concat_axis) * concat_size;

        out_data[top_index] = in_data[index];
    }
}

template <typename dtype>
__global__ void concat_impl_2d_impl(const int inner_size, const int num_concats,
                                    const dtype* in_data, const int concat_size,
                                    const int out_concat_axis,
                                    const int offset_concat_axis, dtype* out_data) {

    int idx_inner = threadIdx.x + blockIdx.x * blockDim.x;
    int idx_outer = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx_inner < inner_size && idx_outer < num_concats) {
        int idx_input = idx_outer * inner_size + idx_inner;
        int idx_output = (idx_outer * out_concat_axis + offset_concat_axis) * \
            concat_size + idx_inner;
        out_data[idx_output] = in_data[idx_input];
    }

}


template <DataType OpDtype ,
    DataType inDtype,
    DataType outDtype,
    typename LayOutType_op,
    typename LayOutType_in,
    typename LayOutType_out>
SaberStatus SaberConcat<NV, OpDtype, inDtype, outDtype,\
    LayOutType_op, LayOutType_in, LayOutType_out>::dispatch(const std::vector<DataTensor_in *>& inputs,
    std::vector<DataTensor_out *>& outputs,
    ConcatParam<OpTensor>& param) {

    hipStream_t stream = this->_ctx->get_compute_stream();

    int input_size = inputs.size();

    #if 0 //disable share memory
    //! inputs only has one tensor
    if (input_size == 1) {
        outputs[0]->set_shape(outputs[0]->valid_shape(), inputs[0]->shape(), \
                inputs[0]->offset());
        outputs[0]->share_from(*inputs[0]);
        return;
    }

    //! check whether the output is shared from input tensors
    bool share_mem = false;
    Shape offset_min = inputs[0]->offset();
    const dtype* ptr = inputs[0]->data();
    for (int i = 1; i < input_size; ++i) {
        const dtype* ptr2= inputs[i]->data();
        if (inputs[i]->offset() < offset_min) {
            offset_min = inputs[i]->offset();
        }
        share_mem = (ptr == ptr2);
        if (!share_mem){
            break;
        }
    }
    //! input tensors are sharing one tensor
    if (share_mem){
        CHECK_LE(outputs[0]->valid_size(), inputs[0]->size()) << "input shared tensors overlap";
        outputs[0]->set_shape(outputs[0]->valid_shape(), inputs[0]->shape(), offset_min);
        outputs[0]->share_from(*inputs[0]);
        return;
    }
    #endif // disable share memory

    //! get output data, valid shape and stride shape
    OutDataType* out_data = outputs[0]->mutable_data();
    int offset_concat_axis = 0;
    Shape out_shape = outputs[0]->valid_shape();
    const int out_concat_axis = out_shape[param.axis];
    bool out_cont_flag = outputs[0]->is_continue_mem();
    bool in_cont_flag = inputs[0]->is_continue_mem();
    for (int i = 1; i < input_size; ++i) {
        in_cont_flag &= inputs[i]->is_continue_mem();
    }

    //! inputs and outputs are all with continuous memory
    if (in_cont_flag && out_cont_flag){
        for (int i = 0; i < input_size; ++i) {
            Shape in_shape = inputs[i]->valid_shape();
            //std::vector<int> bottom_shape = {tmp[3], tmp[2], tmp[1], tmp[0]};
            const InDataType* in_data = inputs[i]->data();
            const int in_concat_axis = in_shape[param.axis];
            const int in_concat_size = in_concat_axis * _concat_input_size;
            const int nthreads = in_concat_size * _num_concats;
            float ratio = (float)in_concat_size / _num_concats;
            bool is_balance = (ratio > 0.1 && ratio < 10);
            if (is_balance){
                int block_x = BLOCK_SIZE;
                int block_y = BLOCK_SIZE;
                int grid_x = (in_concat_size + block_x - 1) / block_x;
                int grid_y = (_num_concats + block_y - 1) / block_y;
                dim3 block(block_x, block_y);
                dim3 grid(grid_x, grid_y);
                concat_impl_2d_impl<InDataType><<<grid, block, 0, stream>>>(
                        in_concat_size, _num_concats, in_data, _concat_input_size,
                                out_concat_axis, offset_concat_axis, out_data
                );
            } else {
                // NOLINT_NEXT_LINE(whitespace/operators)
                concat_impl_cuda<InDataType><<<CUDA_GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, stream>>>( \
                    nthreads, in_data, _num_concats, _concat_input_size, \
                            out_concat_axis, in_concat_axis, offset_concat_axis, out_data);
            }
            offset_concat_axis += in_concat_axis;
        }
    } else { //! inputs or outputs memory is not continuous
#if 1
        Shape offset_out = outputs[0]->offset();
        OpTensor tsub;
        for (int i = 0; i < input_size; ++i) {
            Shape in_shape = inputs[i]->valid_shape();
            tsub.share_sub_buffer(*outputs[0], in_shape, offset_out);
            offset_out[param.axis] += in_shape[param.axis];
            tsub.async_copy_from(*inputs[i], stream);
        }
#endif
    }

    //outputs[0]->record_event(stream);
    return SaberSuccess;
}
#if 0
typedef Tensor<NV, AK_FLOAT, NCHW> Tensor4f_1;
typedef Tensor<NV, AK_FLOAT, NHWC> Tensor4f_2;
typedef Tensor<NV, AK_FLOAT, HW> Tensor2f;
typedef Tensor<NV, AK_INT8, NCHW> Tensor4c_1;
typedef Tensor<NV, AK_INT8, NHWC> Tensor4c_2;
typedef Tensor<NV, AK_INT8, HW> Tensor2c;
template SaberStatus SaberConcat<AK_FLOAT, NCHW>::dispatch(const std::vector<Tensor4f_1 *> inputs, std::vector<Tensor4f_1 *> outputs,
                                              ConcatParam<Tensor4f_1> &param);
template SaberStatus SaberConcat<AK_FLOAT, NHWC>::dispatch(const std::vector<Tensor4f_2 *> inputs, std::vector<Tensor4f_2 *> outputs,
                                              ConcatParam<Tensor4f_2> &param);
template SaberStatus SaberConcat<AK_FLOAT, HW>::dispatch(const std::vector<Tensor2f *> inputs, std::vector<Tensor2f *> outputs,
                                              ConcatParam<Tensor2f> &param);
template SaberStatus SaberConcat<AK_INT8, NCHW>::dispatch(const std::vector<Tensor4c_1 *> inputs, std::vector<Tensor4c_1 *> outputs,
                                              ConcatParam<Tensor4c_1> &param);
template SaberStatus SaberConcat<AK_INT8, NHWC>::dispatch(const std::vector<Tensor4c_2 *> inputs, std::vector<Tensor4c_2 *> outputs,
                                                   ConcatParam<Tensor4c_2> &param);
template SaberStatus SaberConcat<AK_INT8, HW>::dispatch(const std::vector<Tensor2c *> inputs, std::vector<Tensor2c *> outputs,
                                                   ConcatParam<Tensor2c> &param);
#endif
} //namespace anakin

} //namespace anakin
