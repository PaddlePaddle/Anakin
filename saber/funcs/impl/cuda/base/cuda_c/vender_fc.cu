#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/vender_fc.h"


namespace anakin{

namespace saber{

template <typename dtype>
void anakin_NV_gemv(hipblasHandle_t handle, const bool TransA, \
					 const int M, const int N, \
					 const dtype alpha, const dtype* A,\
					 const dtype* x, const dtype beta,\
					 dtype* y);

template <>
void anakin_NV_gemv<float>(hipblasHandle_t handle, const bool TransA, \
    const int M, const int N, const float alpha, const float* A, const float* x, \
    const float beta, float* y) {
    hipblasOperation_t cuTransA = (TransA == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        CUBLAS_CHECK(hipblasSgemv(handle, cuTransA, N, M, &alpha,
                                 A, N, x, 1, &beta, y, 1));
    }

template <>
void anakin_NV_gemv<char>(hipblasHandle_t handle, const bool TransA, \
					 const int M, const int N, \
					 const char alpha, const char* A,\
					 const char* x, const char beta,\
					 char* y) {
    LOG(FATAL) << "int8 gemv is not implemented";
}

template <typename dtype>
void anakin_NV_gemm(hipblasHandle_t handle, const bool TransA,
                    const bool TransB, const int M, const int N, const int K,
                    const dtype alpha, const dtype* A, const dtype* B, const dtype beta,
                    dtype* C);

template <>
void anakin_NV_gemm<float>(hipblasHandle_t handle, const bool TransA,
                           const bool TransB, const int M, const int N, const int K,
                           const float alpha, const float* A, const float* B, const float beta,
                           float* C) {
    // Note that cublas follows fortran order.
    int lda = (!TransA/* == CblasNoTrans*/) ? K : M;
    int ldb = (!TransB/* == CblasNoTrans*/) ? N : K;
    hipblasOperation_t cuTransA =
            (!TransA/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
            (!TransB/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA,
                             N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void anakin_NV_gemm<char>(hipblasHandle_t handle, const bool TransA,
                           const bool TransB, const int M, const int N, const int K,
                           const char alpha, const char* A, const char* B, const char beta,
                           char* C) {
    LOG(FATAL) << "int8 gemm is not implemented";
}

template <typename dtype>
__global__ void add_bias(int n, int output_size, const dtype* bias, dtype* dout) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int bias_index = index % output_size;
    if (index < n) {
        //printf("index: %d, bias_index: %d, val_in: %.2f\n", index, bias_index, bias[bias_index]);
        dout[index] = dout[index] + bias[bias_index];
    }
}

template <DataType OpDtype,
        DataType inDtype,
        DataType outDtype,
        typename LayOutType_op,
        typename LayOutType_in,
        typename LayOutType_out>
SaberStatus VenderFc<NV, OpDtype, inDtype, outDtype, \
    LayOutType_op, LayOutType_in, LayOutType_out>::dispatch(
            const std::vector<DataTensor_in *>& inputs,
            std::vector<DataTensor_out *>& outputs,
            FcParam<OpTensor>& param) {

    hipStream_t stream = this->_ctx->get_compute_stream();

    const InDataType* din = inputs[0]->data();
    OutDataType* dout = outputs[0]->mutable_data();
    const OpDataType* weight = param.weights->data();
    const InDataType* bias = nullptr;
    bool bias_term = param.bias != nullptr;
    //dim3 grid(CUDA_GET_BLOCKS(param.num_output), _M);
    if (bias_term) {
        bias = param.bias->data();
    }

    if (_M == 1 && _K > 50000) {
        anakin_NV_gemv<InDataType>(_handle, false, _N, _K, (InDataType)1, weight, din, \
            (InDataType)0, dout);
    } else {
        anakin_NV_gemm<InDataType>(_handle, false, !_flag_trans_weights, \
            _M, _N, _K, (InDataType)1, din, weight, (InDataType)0, dout);
    }
    if (bias_term) {
        int total_size = _M * _N;
        add_bias<InDataType><<<CUDA_GET_BLOCKS(total_size), CUDA_NUM_THREADS, 0, stream>>>\
            (total_size, _N, bias, dout);
    }
    return SaberSuccess;
}

/*
#define INSTANCE_FC(datatype, layouttype) \
    template SaberStatus CublasFc<datatype, layouttype>::dispatch( \
        const std::vector<CublasFc<datatype, layouttype>::ioTensor *> inputs, \
        std::vector<CublasFc<datatype, layouttype>::ioTensor *> outputs, \
        FcParam<CublasFc<datatype, layouttype>::ioTensor> &param);

INSTANCE_FC(AK_FLOAT, NCHW);
INSTANCE_FC(AK_INT8, NCHW);
INSTANCE_FC(AK_FLOAT, NHWC);
INSTANCE_FC(AK_INT8, NHWC);
*/
} //namespace anakin

} //namespace anakin
