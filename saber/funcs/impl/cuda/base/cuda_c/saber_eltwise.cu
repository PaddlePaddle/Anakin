#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_eltwise.h"
namespace anakin {
namespace saber {

template <typename Dtype>
__global__ void ker_elt_production(Dtype* out_data, const Dtype * in_data_a, const Dtype * in_data_b, int count){
    CUDA_KERNEL_LOOP(tid, count){
        out_data[tid] = in_data_a[tid] * in_data_b[tid];
    }
}

template <typename Dtype>
__global__ void ker_elt_sum(Dtype* out_data, const Dtype * in_data1,const Dtype * in_data2, Dtype coeff1,  Dtype coeff2, int count){
    CUDA_KERNEL_LOOP(tid, count){
        out_data[tid] = coeff1*in_data1[tid] + coeff2 * in_data2[tid];
    }
}

template <typename Dtype>
__global__ void ker_elt_max(Dtype * out_data, float * mask, const Dtype * in_data_a, const Dtype * in_data_b, int count, int bid){
    if(bid == 0){
        CUDA_KERNEL_LOOP(tid, count){
            Dtype var_a = in_data_a[tid];
            Dtype var_b = in_data_b[tid];
            bool a_gt_b = var_a > var_b;
            out_data[tid] = a_gt_b ? var_a : var_b;
            mask[tid] = a_gt_b ? 0 : 1;
        }
    }
    else{
        CUDA_KERNEL_LOOP(tid, count){
            Dtype var_a = in_data_a[tid];
            Dtype var_b = in_data_b[tid];
            bool a_gt_b = var_a > var_b;
            if( ! a_gt_b){
                out_data[tid] = var_b;
                mask[tid] = bid;
            }
        }
    }
}


template <>
SaberStatus SaberEltwise<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
    const std::vector<DataTensor_in*>& inputs, \
    std::vector<DataTensor_out*>& outputs, \
    EltwiseParam<OpTensor> &param) {
    float * mask = NULL;
    const int count = outputs[0]->size();
    float *out_data = outputs[0]->mutable_data();
    const float *in_data_a = inputs[0]->data();
	const float *in_data_b = inputs[1]->data();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    switch(param.operation){
	case Eltwise_prod:
		ker_elt_production<InDataType>
            <<<CUDA_GET_BLOCKS(count),
            CUDA_NUM_THREADS, 0, hip_stream>>>(out_data, in_data_a,
                    in_data_b, count);

		for(int i = 2; i < inputs.size(); i++){
			ker_elt_production<InDataType>
            <<<CUDA_GET_BLOCKS(count),
            CUDA_NUM_THREADS, 0, hip_stream>>>(out_data, out_data,
                    inputs[i]->data(), count);
		}
		break;
	case Eltwise_sum:
		ker_elt_sum<InDataType>
            <<<CUDA_GET_BLOCKS(count),
            CUDA_NUM_THREADS, 0, hip_stream>>>(out_data,
                    inputs[0]->data(), inputs[1]->data(),
                    param.coeff[0], param.coeff[1], count);
		break;
	case Eltwise_max:
		mask = _max_idx.mutable_data();
		ker_elt_max<InDataType>
            <<<CUDA_GET_BLOCKS(count),
            CUDA_NUM_THREADS, 0, hip_stream>>>(out_data, mask,
                    in_data_a, in_data_b, count, 0);

		for(int i = 2; i < inputs.size(); i++){
			ker_elt_max<InDataType>
            <<<CUDA_GET_BLOCKS(count),
            CUDA_NUM_THREADS, 0, hip_stream>>>(out_data, mask,
                    out_data, inputs[i]->data(), count, i);
		}
		break;
	default:
		LOG(FATAL) << "unknown elementwise operation. ";
	}

    CUDA_POST_KERNEL_CHECK;
    return SaberSuccess;
}

}
}