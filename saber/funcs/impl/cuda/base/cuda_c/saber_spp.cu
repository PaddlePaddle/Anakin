#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_spp.h"
#include "hip/hip_fp16.h"

namespace anakin {

namespace saber {

#if 0    
template <typename Dtype>
__global__ void ker_concat_fwd(Dtype* out_data, const Dtype* in_data,
                               const int n,
                               const int w,
                               const int n_stride, const int nthreads) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int n_id = index / w;
        const int w_id = index % w;
        const int out_index = n_id * n_stride + w_id;
        out_data[out_index] = in_data[index];
    }
}

template <DataType OpDtype,
            DataType inDtype,
            DataType outDtype,
            typename LayOutType_op,
            typename LayOutType_in,
            typename LayOutType_out>
SaberStatus SaberSpp<NV, OpDtype, inDtype, outDtype,\
    LayOutType_op, LayOutType_in, LayOutType_out>::dispatch(\
    const std::vector<DataTensor_in *>& inputs, \
    std::vector<DataTensor_out *>& outputs, \
    SPPParam<OpTensor>& param) {

    const InDataType* in_data = inputs[0]->data();
    OutDataType* out_data = outputs[0]->mutable_data();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    int count = outputs[0]->valid_size();
    int out_n = outputs[0]->num();
    int out_c = outputs[0]->channel();
    int out_h = outputs[0]->height();
    int out_w = outputs[0]->width();

    if (inputs[0]->is_continue_mem() && outputs[0]->is_continue_mem()) {
        std::vector<OpTensor*> pool_outputs;
        pool_outputs.resize(1);
        for (int i = 0; i < param.pyramid_height; i++) {
            pool_outputs[0] = _pooling_output[i];
            (*_pooling[i])(inputs, pool_outputs, _pooling_param[i], this->_ctx);
            int valid_size  = pool_outputs[0]->valid_size();
            int offset = (pow(4, i) - 1) / 3;
            ker_concat_fwd<InDataType><<<CUDA_GET_BLOCKS(valid_size),CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data + offset, 
                    pool_outputs[0]->data(), 
                    pool_outputs[0]->num() * pool_outputs[0]->channel(), 
                    pool_outputs[0]->height() * pool_outputs[0]->width(), 
                    outputs[0]->width(), 
                    valid_size);
        }
    }

    return SaberSuccess;
}
#endif
} //namespace saber

} //namespace anakin
