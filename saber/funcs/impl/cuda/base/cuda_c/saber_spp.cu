#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_spp.h"
#include "saber/core/tensor_op.h"
#include "hip/hip_fp16.h"

namespace anakin {

namespace saber {

template <typename Dtype>
__global__ void ker_concat_fwd(Dtype* out_data, const Dtype* in_data,
                               const int n,
                               const int w,
                               const int n_stride, const int nthreads) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int n_id = index / w;
        const int w_id = index % w;
        const int out_index = n_id * n_stride + w_id;
        out_data[out_index] = in_data[index];
    }
}

template <DataType OpDtype>
SaberStatus SaberSpp<NV, OpDtype>::dispatch(\
    const std::vector<DataTensor_in *>& inputs, \
    std::vector<DataTensor_out *>& outputs, \
    SPPParam<NV>& param) {

    const InDataType* in_data = (const InDataType*)inputs[0]->data();
    OutDataType* out_data = (OutDataType*)outputs[0]->mutable_data();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    int count = outputs[0]->valid_size();
    int out_n = outputs[0]->num();
    int out_c = outputs[0]->channel();
    int out_h = outputs[0]->height();
    int out_w = outputs[0]->width();

    if (inputs[0]->is_continue_mem() && outputs[0]->is_continue_mem()) {
        std::vector<OpTensor*> pool_outputs;
        pool_outputs.resize(1);
        for (int i = 0; i < param.pyramid_height; i++) {
            pool_outputs[0] = _pooling_output[i];
            (*_pooling[i])(inputs, pool_outputs, _pooling_param[i], *(this->_ctx));
            int valid_size  = pool_outputs[0]->valid_size();
            int offset = (pow(4, i) - 1) / 3;
            ker_concat_fwd<InDataType><<<CUDA_GET_BLOCKS(valid_size),CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data + offset, 
                    (InDataType*) pool_outputs[0]->data(), 
                    pool_outputs[0]->num() * pool_outputs[0]->channel(), 
                    pool_outputs[0]->height() * pool_outputs[0]->width(), 
                    outputs[0]->width(), 
                    valid_size);
        }
    }

    return SaberSuccess;
}
} //namespace saber

} //namespace anakin
