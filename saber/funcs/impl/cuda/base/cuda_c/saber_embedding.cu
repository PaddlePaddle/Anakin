#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_embedding.h"
#include "hip/hip_fp16.h"

namespace anakin{
namespace saber{

template<typename InDataType, typename OpDataType, typename OutDataType>
__global__ void ker_embedding_fwd(OutDataType * out_data,
                                const InDataType* in_data,
                                const OpDataType* tabel,
                                const int total_word_num,
                                const int emb_dim,
                                const int word_num,
                                const int padding_idx,
                                const int out_count) {

    CUDA_KERNEL_LOOP(tid, out_count){
        int emb_id =  tid % emb_dim;
        int word_id = tid / emb_dim;
        int word_idx_in_tabel = (int)(in_data[word_id]);
        if (word_idx_in_tabel != padding_idx) {
            out_data[tid] = OutDataType(tabel[word_idx_in_tabel * emb_dim + emb_id]);
        } else {
            out_data[tid] = OutDataType(0.f);
        }
    }
}

template <DataType OpDtype>
SaberStatus SaberEmbedding<NV, OpDtype>::dispatch( \
    const std::vector<Tensor<NV>*>& inputs,
    std::vector<Tensor<NV>*>& outputs,
	EmbeddingParam<NV>& param) {

    CHECK_EQ(inputs[0]->get_dtype(), AK_FLOAT) <<" Embedding only support float inputs.";
    const OpDataType *op_data = (const OpDataType*)(param.weight()->data());

    const int count = outputs[0]->valid_size();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    
    //outputs: chose corresponding informations of words.
    //inputs: word_id [Its type maybe float or int]
    //outputs = weights[inputs[j]].
    ker_embedding_fwd<float, OpDataType, OpDataType>
    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                (OpDataType*)outputs[0]->mutable_data(), 
                (const float*)inputs[0]->data(), op_data, 
                param.word_num, param.emb_dim, inputs[0]->num(),
                param.padding_idx, outputs[0]->valid_size());
    
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    CUDA_POST_KERNEL_CHECK;
    return SaberSuccess;
}

template class SaberEmbedding<NV, AK_FLOAT>;
template class SaberEmbedding<NV, AK_INT8>;
DEFINE_OP_TEMPLATE(SaberEmbedding, EmbeddingParam, NV, AK_HALF);
}
}