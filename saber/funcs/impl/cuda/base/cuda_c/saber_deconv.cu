#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_deconv.h"
#include "saber/funcs/saber_util.h"

namespace anakin {

namespace saber {

template <typename Dtype, bool with_relu>
static __global__ void ker_bias_relu(Dtype* tensor, const Dtype* bias, int channel_num,
                                     int channel_size) {
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int channel_id = thread_id / channel_size;
    const int channel_inner_index = thread_id % channel_size;

    if (channel_id < channel_num) {
        Dtype tmp = tensor[thread_id] + bias[channel_id];

        if (with_relu) {
            tensor[thread_id] = tmp > 0 ? tmp : 0;
        } else {
            tensor[thread_id] = tmp;
        }
    }
};

template <typename Dtype>
static inline void bias_relu(Dtype* tensor, const Dtype* bias, int channel_num, int channel_size,
                             int with_relu, hipStream_t stream) {
    if (with_relu) {
        ker_bias_relu<Dtype, true> <<< CUDA_GET_BLOCKS(channel_num* channel_size),
                      CUDA_NUM_THREADS, 0, stream>>>(tensor, bias, channel_num, channel_size);
    } else {
        ker_bias_relu<Dtype, true> <<< CUDA_GET_BLOCKS(channel_num* channel_size),
                      CUDA_NUM_THREADS, 0, stream>>>(tensor, bias, channel_num, channel_size);
    }
}

template <typename Dtype>
static __global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
        const int height, const int width, const int channels,
        const int kernel_h, const int kernel_w,
        const int pad_h, const int pad_w,
        const int stride_h, const int stride_w,
        const int dilation_h, const int dilation_w,
        const int height_col, const int width_col,
        Dtype* data_im) {
    CUDA_KERNEL_LOOP(index, n) {
        Dtype val = 0;
        const int w_im = index % width + pad_w;
        const int h_im = (index / width) % height + pad_h;
        const int c_im = index / (width * height);
        int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
        int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
        // compute the start and end of the output
        const int w_col_start =
            (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
        const int w_col_end = min(w_im / stride_w + 1, width_col);
        const int h_col_start =
            (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
        const int h_col_end = min(h_im / stride_h + 1, height_col);

        // TODO: use LCM of stride and dilation to avoid unnecessary loops
        for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
            for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
                int h_k = (h_im - h_col * stride_h);
                int w_k = (w_im - w_col * stride_w);

                if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
                    h_k /= dilation_h;
                    w_k /= dilation_w;
                    int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                          height_col + h_col) * width_col + w_col;
                    val += data_col[data_col_index];
                }
            }
        }

        data_im[index] = val;
    }
}

template <typename Dtype>
static void col2im_gpu(const Dtype* data_col, const int channels,
                       const int height, const int width, const int kernel_h, const int kernel_w,
                       const int pad_h, const int pad_w, const int stride_h,
                       const int stride_w, const int dilation_h, const int dilation_w,
                       Dtype* data_im, hipStream_t stream) {
    int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
                     stride_h + 1;
    int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
                    stride_w + 1;
    int num_kernels = channels * height * width;
    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    // NOLINT_NEXT_LINE(whitespace/operators)
    col2im_gpu_kernel<Dtype> <<< CUDA_GET_BLOCKS(num_kernels),
                      CUDA_NUM_THREADS, 0, stream>>>(
                          num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
                          pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
                          height_col, width_col, data_im);
}

template <typename dtype, bool flag_bias, bool flag_act>
__global__ void direct_deconv(const dtype* const din,
                              const dtype* bias_data, const dtype* const weight_data,
                              const int num, const int in_channels, const int out_channels,
                              const int hout, const int wout, const int channel_out_stride,
                              const int hin, const int win, const int channel_in_stride,
                              const int kernel_h, const int kernel_w, const int kernel_size,
                              const int stride_h, const int stride_w,
                              const int pad_h, const int pad_w,
                              const int dilation_h, const int dilation_w,
                              dtype* dout) {

    int wo = blockIdx.x * blockDim.x + threadIdx.x;
    int w =  wo + pad_w;
    int ho = blockIdx.y * blockDim.y + threadIdx.y;
    int h =  ho + pad_h;
    int iout = blockIdx.z;
    int cout = iout % out_channels;
    int n = iout / out_channels;
    int iin = n * in_channels;
    int idx_out = iout * channel_out_stride + ho * wout + wo;

    extern __shared__ dtype sharedw[];
    dtype val = 0;

    if (wo < wout && ho < hout) {
        for (int ic = 0; ic < in_channels; ic++) {
            //! read weights
            int idx_weight = threadIdx.y * blockDim.x + threadIdx.x;

            if (idx_weight < kernel_size) {
                sharedw[idx_weight] = weight_data[(ic * out_channels + cout) * kernel_size + idx_weight];
            }

            __syncthreads();
            //! get start and end index
            const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
            const int phend = min(h / stride_h + 1, hin);
            const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
            const int pwend = min(w / stride_w + 1, win);

            const int khstart = (h >= kernel_h) ? ((h - kernel_h) % stride_h) + (kernel_h - stride_h) : h;
            const int kwstart = (w >= kernel_w) ? ((w - kernel_w) % stride_w) + (kernel_w - stride_w) : w;

            const dtype* const din_c = din + (iin + ic) * channel_in_stride;

            //! start computation
            for (int ph = phstart; ph < phend; ++ph) {
                for (int pw = pwstart; pw < pwend; ++pw) {
                    int kh = khstart - (ph - phstart) * stride_h;
                    int kw = kwstart - (pw - pwstart) * stride_w;
                    val += din_c[ph * win + pw] * sharedw[kh * kernel_w + kw];
                }
            }
        }
        //! finnal computation
        if (flag_bias) {
            val += bias_data[cout];
        }
        if (flag_act) {
            val = val > (dtype)0 ? val : (dtype)0;
        }
        dout[idx_out] = val;
    }
}

template <typename dtype, bool bias_flag, bool relu_flag>
__global__ void depthwise_deconv_2d(const int channel_in_stride, const int channel_out_stride,
                                    const int kernel_size,
                                    const dtype* const din, const int num, const int channels,
                                    const int hin, const int win, const int hout,
                                    const int wout, const int kernel_h, const int kernel_w,
                                    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
                                    dtype* const dout, const dtype* const weight, const dtype* const bias) {

    int wo = blockIdx.x * blockDim.x + threadIdx.x;
    int w =  wo + pad_w;
    int ho = blockIdx.y * blockDim.y + threadIdx.y;
    int h =  ho + pad_h;
    int c = blockIdx.z % channels;
    //int n = blockIdx.z / channels;
    int i = blockIdx.z;
    int index = i * channel_out_stride + ho * wout + wo;

    extern __shared__ dtype sharedw[];
    int idx = threadIdx.y * blockDim.x + threadIdx.x;

    if (idx < kernel_size) {
        sharedw[idx] = weight[c * kernel_size + idx];
    }
    __syncthreads();

    if (wo < wout && ho < hout) {
        const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
        const int phend = min(h / stride_h + 1, hin);
        const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
        const int pwend = min(w / stride_w + 1, win);

        const int khstart = (h >= kernel_h) ? ((h - kernel_h) % stride_h) + (kernel_h - stride_h) : h;
        const int kwstart = (w >= kernel_w) ? ((w - kernel_w) % stride_w) + (kernel_w - stride_w) : w;

        dtype gradient = 0;
        const dtype* const top_diff_slice = din + i * channel_in_stride;
        const dtype* const weight_slice = weight + c * kernel_size;

        for (int ph = phstart; ph < phend; ++ph) {
            for (int pw = pwstart; pw < pwend; ++pw) {
                int kh = khstart - (ph - phstart) * stride_h;
                int kw = kwstart - (pw - pwstart) * stride_w;
                gradient += top_diff_slice[ph * win + pw] * sharedw[kh * kernel_w + kw];
                //gradient += top_diff_slice[ph * win + pw] * weight_slice[kh * kernel_w + kw];
            }
        }
        if (bias_flag) {
            gradient += bias[c];
        }

        if (relu_flag) {
            gradient = gradient > (dtype)0 ? gradient : (dtype)0;
        }

        dout[index] = gradient;
    }
}


template <>
SaberStatus SaberDeconv2D<NV, AK_FLOAT>::create(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        ConvParam<NV>& param, Context<NV> &ctx) {
    _use_k4_s2_p1 = true;
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.weight()->width() == 4);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.weight()->height() == 4);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.stride_h == 2);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.stride_w == 2);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.pad_h == 1);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.pad_w == 1);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (param.group == 1);
    _use_k4_s2_p1 = _use_k4_s2_p1 && (inputs[0]->width() % 64 == 0);
    if (_use_k4_s2_p1) {
        int in_channel = inputs[0]->channel();
        int out_channel = outputs[0]->channel();
        scale_to_new_tensor_k4_s2_p1_deconv<4>(param.mutable_weight(),
                                               in_channel, out_channel);
        return SaberSuccess;
    } else {
        return SaberUnImplError;
    }
}

template <>
SaberStatus SaberDeconv2D<NV, AK_FLOAT>::init(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        ConvParam<NV>& param, Context<NV>& ctx) {
    this->_ctx = &ctx;
    return create(inputs, outputs, param, ctx);
}

template <>
SaberStatus SaberDeconv2D<NV, AK_FLOAT>::dispatch(\
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        ConvParam<NV>& param) {
    hipStream_t stream = this->_ctx->get_compute_stream();

    const float* din = (const float*)inputs[0]->data();
    float* dout = (float*)outputs[0]->mutable_data();
    const float* weight = (const float*)param.weight()->data();

    int win = inputs[0]->width();
    int hin = inputs[0]->height();
    int num = inputs[0]->num();
    int ch_in = inputs[0]->channel();
    int wout = outputs[0]->width();
    int hout = outputs[0]->height();
    int ch_out = outputs[0]->channel();

    int kernel_w = param.weight()->width();
    int kernel_h = param.weight()->height();

    if (_use_k4_s2_p1) {
        const float * bias_data = (param.bias()->valid_size() > 0) ?
                                  (const float*)param.bias()->data() : NULL;
        const float *weights_data = (const float*)param.weight()->data();
        ker_deconv_implicit_gemm_k4_s2_p1_16x64(dout, din,
                                                weights_data, bias_data,
                                                num,
                                                hin, win, hout, wout,
                                                ch_in, ch_out, stream);
        return SaberSuccess;
    } else {
        return SaberUnImplError;
    }
}
template class SaberDeconv2D<NV, AK_FLOAT>;
DEFINE_OP_TEMPLATE(SaberDeconv2D, ConvParam, NV, AK_HALF);
DEFINE_OP_TEMPLATE(SaberDeconv2D, ConvParam, NV, AK_INT8);
} //namespace anakin

} //namespace anakin
