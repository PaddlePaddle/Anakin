#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_crf_decoding.h"
#include "hip/hip_fp16.h"


namespace anakin{

namespace saber{

template<typename Dtype, unsigned int blockSize>
__global__ void decoding_kernel2(Dtype* decode_path, const Dtype* emission_ptr, const Dtype* trans_ptr, \
                    Dtype* alpha_ptr, int* track_ptr, int* seq_offset, int seq_num, int slice_size, int tag_num, const int base_idx){

    int bdx = blockIdx.x;
    if (bdx >= seq_num){
        return;
    }
    int seq_len = seq_offset[bdx];
    int sum = 0;
    int sum2 = 0;
    for (int i = 0; i < bdx; i++){
        int tmp = seq_offset[i];
        sum += tmp;
        sum2 += tmp * slice_size;
    }
    Dtype* path = decode_path + sum;
    const Dtype* emission = emission_ptr + sum2;

    int idx = threadIdx.x;
    const Dtype* x = emission;
    const Dtype* w = trans_ptr;
    if (idx < tag_num){
        alpha_ptr[idx] = trans_ptr[idx] + emission_ptr[idx];
    }
    for (int k = 1; k < seq_len; ++k) {
        if (idx < tag_num) {
            Dtype max_score = -1e32;//-std::numeric_limits<Dtype>::max();
            int max_j = 0;
            for (int j = 0; j < tag_num; ++j) {
                Dtype score = alpha_ptr[(k - 1) * tag_num + j] +
                    w[(j + base_idx) * tag_num + idx];
                if (score > max_score) {
                    max_score = score;
                    max_j = j;
                }
            }
            alpha_ptr[k * tag_num + idx] = max_score + x[k * tag_num + idx];
            track_ptr[k * tag_num + idx] = max_j;
        }
    }
    __syncthreads();
//only run block times
    Dtype max_score = -1e32;
    int max_i = 0;
    for (int i = 0; i < tag_num; i++) {
        Dtype score = alpha_ptr[(seq_len - 1) * tag_num + i] + w[tag_num + i];
        if (score > max_score) {
            max_score = score;
            max_i = i;
        }
    }
    path[seq_len - 1] = max_i;
    for (int k = seq_len - 1; k >= 1; k--) {
        max_i = track_ptr[k * tag_num + max_i];
        path[k - 1] = max_i;
    }
}

template<typename Dtype, unsigned int blockSize>
__global__ void decoding_kernel(Dtype* decode_path, const Dtype* emission_ptr, const Dtype* trans_ptr, \
                    Dtype* alpha_ptr, int* track_ptr, int seq_len, int tag_num, const int base_idx){
    int idx = threadIdx.x;
    const Dtype* x = emission_ptr;
    const Dtype* w = trans_ptr;
    Dtype* alpha_value = alpha_ptr;

    for (int i = 0; i < tag_num; ++i) alpha_value[i] = w[i] + x[i];

    for (int k = 1; k < seq_len; ++k) {
        for (int i = 0; i < tag_num; ++i) {
            Dtype max_score = -1e32;//-std::numeric_limits<Dtype>::max();
            int max_j = 0;
            for (int j = 0; j < tag_num; ++j) {
                Dtype score = alpha_value[(k - 1) * tag_num + j] +
                    w[(j + base_idx) * tag_num + i];
                if (score > max_score) {
                    max_score = score;
                    max_j = j;
                }
            }
            alpha_value[k * tag_num + i] = max_score + x[k * tag_num + i];
            track_ptr[k * tag_num + i] = max_j;
        }
    }
    Dtype max_score = -1e32;
    int max_i = 0;
    for (size_t i = 0; i < tag_num; i++) {
        Dtype score = alpha_ptr[(seq_len - 1) * tag_num + i] + trans_ptr[tag_num + i];
        if (score > max_score) {
            max_score = score;
            max_i = i;
        }
    }
    decode_path[seq_len - 1] = max_i;
    for (int k = seq_len - 1; k >= 1; k--) {
        max_i = track_ptr[k * tag_num + max_i];
        decode_path[k - 1] = max_i;
    }
}

template <>
SaberStatus SaberCrfDecoding<NV, AK_FLOAT>::dispatch( \
                        const std::vector<Tensor<NV> *>& inputs,
                        std::vector<Tensor<NV> *>& outputs,
                        CrfDecodingParam<NV>& param){
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    const OpDataType* emission_ptr = (const OpDataType*)inputs[0]->data();
    const OpDataType* trans_ptr = (const OpDataType*)param.mutable_transition_weight()->data();
    OpDataType* decode_path = (OpDataType*)outputs[0]->mutable_data();

    int tag_num = inputs[0]->channel();
    int slice_size = tag_num * inputs[0]->height() * inputs[0]->width();
    std::vector<std::vector<int>> seq_offset = inputs[0]->get_seq_offset();
    int seq_num = seq_offset[0].size() - 1;
    const int base_idx = 2;
    #if 1
    for (int i = 0; i < seq_num; i++){
        int seq_len = seq_offset[0][i+1] - seq_offset[0][i];
        if (seq_len < 1) continue;
        decoding_kernel<OpDataType, 1><<<1, 1, 0, hip_stream>>>(decode_path, \
            emission_ptr, trans_ptr, (OpDataType*)_alpha.mutable_data(), \
            (int*)_track.mutable_data(), seq_len, tag_num, base_idx);

        emission_ptr += slice_size * seq_len;
        decode_path += seq_len;
    }
    #else
    Tensor<NVHX86> seq_host;
    seq_host.re_alloc(Shape({1, 1, 1, seq_num}, Layout_NCHW), AK_INT32);
    _seq.re_alloc(Shape({1, 1, 1, seq_num}, Layout_NCHW), AK_INT32);
    int* seq = (int*)seq_host.mutable_data();
    for (int i = 0; i < seq_num; i++){
        seq[i] = seq_offset[0][i+1] - seq_offset[0][i];
    }
    _seq.copy_from(seq_host);
    decoding_kernel2<OpDataType, CUDA_NUM_THREADS><<<seq_num, tag_num, 0, hip_stream>>>(decode_path, \
            emission_ptr, trans_ptr, (OpDataType*)_alpha.mutable_data(), \
            (int*)_track.mutable_data(), (int*)_seq.mutable_data(), seq_num, slice_size, tag_num, base_idx);
   // delete seq_host;
   #endif
    return SaberSuccess;
}
DEFINE_OP_TEMPLATE(SaberCrfDecoding, CrfDecodingParam, NV, AK_INT8);
DEFINE_OP_TEMPLATE(SaberCrfDecoding, CrfDecodingParam, NV, AK_HALF);
} //namespace anakin

} //namespace anakin
