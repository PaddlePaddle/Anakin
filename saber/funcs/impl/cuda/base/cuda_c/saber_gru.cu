#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_gru.h"
#include "saber/core/tensor_op.h"

namespace anakin {

namespace saber {

////TODO:can try record vector in shared

template <typename Dtype>
__global__ void trans_map2in(Dtype* output, const Dtype* input, const int* map, int count,
                             int lastdim) {
    CUDA_KERNEL_LE(tid, count) {
        int seq = tid / lastdim;
        output[tid] = input[map[seq] * lastdim + tid % lastdim];
//        printf("in %d = %f\n",tid,output[tid]);
    }
}

template <typename Dtype>
__global__ void trans_map2out(Dtype* output, const Dtype* input, const int* map, int count,
                              int lastdim) {
    CUDA_KERNEL_LE(tid, count) {
        int seq = tid / lastdim;
        output[map[seq]*lastdim + tid % lastdim] = input[tid];
//        printf("out %d = %f\n",map[seq]*lastdim + tid % lastdim,output[map[seq]*lastdim + tid % lastdim]);
    }
}

template <typename Dtype>
void trans_map2out_cfunc(const Dtype*  input, Dtype* output, int word_size,int seq_sum, hipStream_t stream,int *dev_map_vec) {
    int count = seq_sum * word_size;
    int block_dim = count;
    int grid_dim = 1;

    if (count > 1024) {
        block_dim = 256;
        grid_dim = (count + block_dim - 1) / block_dim;
    }

    trans_map2out << < grid_dim, block_dim, 0, stream >> > (output, input, dev_map_vec,
            count, word_size);

//    hipDeviceSynchronize();
}

template <typename Dtype>
void trans_map2in_cfunc(const Dtype*  input, Dtype* output, int hidden_size,int seq_sum, hipStream_t stream,int *dev_map_vec) {
    int count = seq_sum * hidden_size;
    int block_dim = count;
    int grid_dim = 1;

    if (count > 1024) {
        block_dim = 256;
        grid_dim = (count + block_dim - 1) / block_dim;
    }

    trans_map2in << < grid_dim, block_dim, 0, stream >> > (output, input, dev_map_vec,
            count, hidden_size);
//    hipDeviceSynchronize();
//    exit(0);
}



template <>
void SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::seq2hw(\
        std::vector<DataTensor_out*> outputs, std::vector<DataTensor_in*> inputs,
        GruParam<OpTensor>& param, int hidden_size,
        void* real_temp_out
                                                                         ) {
    DataTensor_in* din = inputs[0];
    DataTensor_out* dout = outputs[0];
    int wordsize = din->channel();
    std::vector<int> offset_vec = din->get_seq_offset();
    CHECK_GE(offset_vec.size(), 2) << "offset must >=2" ;
    int batch_size = offset_vec.size() - 1;

    int max_len = 0;
    std::vector<int> length_vec;

    if ((void*)(outputs[0]->data()) == real_temp_out) {
        DLOG(INFO) << "not use inner space";
        return;
    }

    const OutDataType* origin = _temp_tensor_out.data();
    OutDataType* target = dout->mutable_data();

    //source is sequence id in seq target is hw id in seq,map is source to target ptr offset
    int seq_sum = offset_vec[batch_size];
    CUDA_CHECK(hipMemcpyAsync(_temp_map_dev.mutable_data(), _temp_map_host.data(), sizeof(int)*seq_sum,
                               hipMemcpyHostToDevice, _ctx->get_compute_stream()));
    int count=seq_sum * hidden_size;
    int block_dim=count;
    int grid_dim=1;
    if(count>1024){
        block_dim=256;
        grid_dim=(count+block_dim-1)/block_dim;
    }
    trans_map2in <<< grid_dim, block_dim, 0, _ctx->get_compute_stream()>>>(target, origin, _temp_map_dev.data(),
            count, hidden_size);

//    trans_map2in_old <<< 4, 128, 0, _ctx.get_compute_stream()>>>(target, origin, _temp_map_dev.data(),
//            count, hidden_size);

}


//TODO:gem by self, flatten by time, padding by nothing (zhangs)
template <>
const float* SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::hw2seq(\
        std::vector<DataTensor_in*> inputs, GruParam<OpTensor>& param, \
        int word_size, int hidden_size, int& sequence_len) {
    DataTensor_in* din = inputs[0];

    std::vector<int> offset_vec = din->get_seq_offset();
    CHECK_GE(offset_vec.size(), 2) << "offset must >=2" ;
    int batch_size = offset_vec.size() - 1;
    int seq_sum = offset_vec[offset_vec.size() - 1];
    int wordsize = din->channel();
    int max_len = 0;
    std::vector<int> length_vec(batch_size);

    for (int i = 0; i < offset_vec.size() - 1; ++i) {
        int len = offset_vec[i + 1] - offset_vec[i];
        max_len = max_len > len ? max_len : len;
        length_vec[i] = len;
    }

    Shape seq_shape(1, max_len, batch_size, word_size);
    _temp_tensor_in.try_expand_size(seq_shape);

    Shape seq_out_shape(1, max_len, batch_size, hidden_size);
    _temp_tensor_out.try_expand_size(seq_out_shape);

    sequence_len = max_len;

    if (batch_size == 1 || max_len == 1) {
        return din->mutable_data();
    }

    InDataType* target = _temp_tensor_in.mutable_data();
    const InDataType* origin = din->data();

    _temp_map_host.try_expand_size(seq_sum);
    _temp_map_dev.try_expand_size(seq_sum);
    int* map = _temp_map_host.mutable_data();

    if (param.is_reverse) {
        for (int batchid = 0; batchid < batch_size; ++batchid) {
            int batch_offset = max_len - length_vec[batchid];

            for (int seqid = 0; seqid < length_vec[batchid]; ++seqid) {
                int source = (offset_vec[batchid] + seqid);
                int target = ((seqid + batch_offset) * batch_size + batchid);
                map[source] = target;
            }
        }
    } else {
        for (int batchid = 0; batchid < batch_size; ++batchid) {
            for (int seqid = 0; seqid < length_vec[batchid]; ++seqid) {
                int source = (offset_vec[batchid] + seqid);
                int target = (seqid * batch_size + batchid);
                map[source] = target;
            }
        }
    }

    CUDA_CHECK(hipMemcpyAsync(_temp_map_dev.mutable_data(), _temp_map_host.data(), sizeof(int)*seq_sum,
                               hipMemcpyHostToDevice, _ctx->get_compute_stream()));
    int count=seq_sum * wordsize;
    int block_dim=count;
    int grid_dim=1;
    if(count>1024){
        block_dim=256;
        grid_dim=(count+block_dim-1)/block_dim;
    }
    trans_map2out <<< grid_dim, block_dim, 0, _ctx->get_compute_stream()>>>(target, origin, _temp_map_dev.data(),
            count, wordsize);

//    trans_map2out_old <<< 4, 128, 0, _ctx.get_compute_stream()>>>(target, origin, _temp_map_dev.data(),
//            count, wordsize);


    return _temp_tensor_in.data();
}

#define SIGMOID_THRESHOLD_MIN_PADDLE -40.0
#define SIGMOID_THRESHOLD_MAX_PADDLE 13.0
#define EXP_MAX_INPUT_PADDLE 40.0

template <typename Dtype>
 static  __device__ Dtype invalidact(Dtype a) {
            printf("invalid act\n");
}

template <typename Dtype>
 static  __device__ Dtype sigmoid(const Dtype a) {
    return static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-a));
}


template <typename Dtype>
 static __device__ Dtype tanh(const Dtype a) {
    Dtype tmp = -2.0 * a;
    return (2.0 / (1.0 + expf(tmp))) - 1.0;
}

template <typename Dtype>
  static __device__ Dtype identity(const Dtype a) {
    return a;
}

template <typename Dtype>
 static __device__ Dtype relu(const Dtype a) {
    return a > static_cast<Dtype>(0.0) ? a : static_cast<Dtype>(0.0);
}

template <typename Dtype>
 static __device__ Dtype sigmoid_fluid(const Dtype a) {
    const Dtype min = SIGMOID_THRESHOLD_MIN_PADDLE;
    const Dtype max = SIGMOID_THRESHOLD_MAX_PADDLE;
    Dtype tmp = (a < min) ? min : ((a > max) ? max : a);

    return static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-tmp));
}

template <typename Dtype>
 static __device__ Dtype tanh_fluid(const Dtype a) {
    Dtype tmp = -2.0 * a;
    tmp = (tmp > EXP_MAX_INPUT_PADDLE) ? EXP_MAX_INPUT_PADDLE : tmp;
    return (2.0 / (1.0 + expf(tmp))) - 1.0;
}

static float (*act_funcs_cu[])(float)= {&invalidact, &sigmoid, &relu, &tanh, &invalidact, \
                                & invalidact, &identity, &sigmoid_fluid, &tanh_fluid};

static void anakin_NV_gemm(hipblasHandle_t handle, const bool TransA,
                           const bool TransB, const int M, const int N, const int K,
                           const float alpha, const float* A, const float* B, const float beta,
                           float* C) {
    // Note that cublas follows fortran order.
    int lda = (!TransA/* == CblasNoTrans*/) ? K : M;
    int ldb = (!TransB/* == CblasNoTrans*/) ? N : K;
    hipblasOperation_t cuTransA =
        (!TransA/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        (!TransB/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA,
                             N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <typename Dtype>
__global__ void cal_one_kernel_paddlesigmoid_tanh_cudnn_formula(Dtype* w_x_r, Dtype* w_x_z,
        Dtype* w_x_o,
        Dtype* w_h_r, Dtype* w_h_z, Dtype* w_h_o,
        const Dtype* b_r, const Dtype* b_z, const Dtype* b_o,
        int hidden_size, Dtype* output, const Dtype* hidden_pre) {
    int w_base_index = blockIdx.x * hidden_size * 3;
    int h_base_index = blockIdx.x * hidden_size;
    Dtype* in_w_x_r = w_x_r + w_base_index;
    Dtype* in_w_h_r = w_h_r + w_base_index;
    Dtype* in_w_x_z = w_x_z + w_base_index;
    Dtype* in_w_h_z = w_h_z + w_base_index;
    Dtype* in_w_x_o = w_x_o + w_base_index;
    Dtype* in_w_h_o = w_h_o + w_base_index;
    const Dtype* in_hidden_pre = hidden_pre + h_base_index;
    Dtype* out_output = output + h_base_index;

    for (int index = threadIdx.x; index < hidden_size; index += blockDim.x) {
        const Dtype min = SIGMOID_THRESHOLD_MIN_PADDLE;
        const Dtype max = SIGMOID_THRESHOLD_MAX_PADDLE;

        Dtype before_act_r = in_w_x_r[index] + in_w_h_r[index] + b_r[index];
        before_act_r = (before_act_r < min) ? min : ((before_act_r > max) ? max : before_act_r);
        Dtype act_r = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + exp(-before_act_r));

        Dtype before_act_z = in_w_x_z[index] + in_w_h_z[index] + b_z[index];
        before_act_z = (before_act_z < min) ? min : ((before_act_z > max) ? max : before_act_z);
        Dtype act_z = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + exp(-before_act_z));

        Dtype before_act_h = in_w_x_o[index] + in_w_h_o[index] * act_r
                             + b_o[index];
        before_act_h = (before_act_h > EXP_MAX_INPUT_PADDLE) ? EXP_MAX_INPUT_PADDLE : before_act_h;
        Dtype acted = tanhf(before_act_h);
        out_output[index] = (1 - act_z) * acted + act_z * in_hidden_pre[index];
    }
}

template <typename Dtype>
__global__ void cal_one_kernel_sigmoid_tanh_modi_cudnn_formula(Dtype* w_x_r, Dtype* w_x_z,
        Dtype* w_x_o,
        Dtype* w_h_r, Dtype* w_h_z, Dtype* w_h_o,
        const Dtype* b_r, const Dtype* b_z, const Dtype* b_o,
        int hidden_size, Dtype* output, const Dtype* hidden_pre) {

    int w_base_index = blockIdx.x * hidden_size * 3 + threadIdx.x;
    int h_base_index = blockIdx.x * hidden_size + threadIdx.x;

    for (int index = threadIdx.x; index < hidden_size;
            index += blockDim.x, w_base_index += blockDim.x, h_base_index += blockDim.x) {
        Dtype before_act_r = w_x_r[w_base_index] + w_h_r[w_base_index] + b_r[index];
        Dtype act_r = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-before_act_r));
        Dtype before_act_z = w_x_z[w_base_index] + w_h_z[w_base_index] + b_z[index];
        Dtype act_z = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-before_act_z));
        Dtype before_act_h = w_x_o[w_base_index] + w_h_o[w_base_index] * act_r
                             + b_o[index];
        Dtype acted = tanh(before_act_h);
        output[h_base_index] = (static_cast<Dtype>(1.0) - act_z) * acted + act_z * hidden_pre[h_base_index];
    }
}


#define CAL_KERNEL_DEFINE(GATACTNAME)\
template <typename Dtype>\
__global__ void cal_reset_kernel##GATACTNAME(Dtype* w_x_r,Dtype* w_h_r,const Dtype* b_r,int hidden_size, Dtype* output, const Dtype* hidden_pre) {\
    int index = threadIdx.x;\
    if (index > hidden_size) {\
        return;\
    }\
    int w_base_index = blockIdx.x * hidden_size * 3 + index;\
    int u_base_index = blockIdx.x * hidden_size * 2 + index;\
    int h_base_index = blockIdx.x * hidden_size + index;\
    Dtype hidden_pre_value = hidden_pre[h_base_index];\
    Dtype before_act_r = w_x_r[w_base_index] + w_h_r[u_base_index] + b_r[index];\
    Dtype act_r = GATACTNAME(before_act_r);\
    output[h_base_index] = hidden_pre_value * act_r;\
};


#define FINAL_KERNEL_DEFINE(GATACTNAME,OUTACTNAME)\
template <typename Dtype>\
__global__ void cal_final_kernel##GATACTNAME##OUTACTNAME( Dtype* w_x_z, Dtype* w_x_o,Dtype* w_h_z,const Dtype* b_z, const Dtype* b_o,\
        int hidden_size, Dtype* output, const Dtype* hidden_pre,const Dtype* w_h_o) {\
    int index = threadIdx.x;\
    if (index > hidden_size) {\
        return;\
    }\
\
    int w_base_index = blockIdx.x * hidden_size * 3 + index;\
    int u_base_index = blockIdx.x * hidden_size * 2 + index;\
    int h_base_index = blockIdx.x * hidden_size + index;\
    Dtype hidden_pre_value = hidden_pre[h_base_index];\
    Dtype before_act_z = w_x_z[w_base_index] + w_h_z[u_base_index] + b_z[index];\
    Dtype act_z =  GATACTNAME(before_act_z);\
    Dtype before_act_h = w_x_o[w_base_index] + w_h_o[h_base_index]\
                         + b_o[index];\
    Dtype acted = OUTACTNAME(before_act_h);\
\
    output[h_base_index] = (static_cast<Dtype>(1.0) - act_z) * hidden_pre_value + act_z * acted;\
}

#define RESET_KERNEL_NAME(GATACTNAME) cal_reset_kernel##GATACTNAME
#define FINAL_KERNEL_NAME(GATACTNAME,OUTACTNAME) cal_final_kernel##GATACTNAME##OUTACTNAME

CAL_KERNEL_DEFINE(sigmoid);

CAL_KERNEL_DEFINE(sigmoid_fluid);

FINAL_KERNEL_DEFINE(sigmoid_fluid,tanh_fluid);

FINAL_KERNEL_DEFINE(sigmoid_fluid,relu);


template <>
SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::gru_cudnn(
    const std::vector<DataTensor_in*> inputs,
    std::vector<DataTensor_out*> outputs,
    GruParam<OpTensor>& param) {

    DataTensor_in* x = inputs[0];
    const InDataType* x_data = x->data();
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1;; //x->get_seq_offset().size()-1;
    int sequence = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;


    if (isHW2Seq) {
        x_data = hw2seq(inputs, param, _word_size, hidden_size, sequence);
        batch_size = offset.size() - 1;

        if (x_data != x->data()) {
            dout_data = _temp_tensor_out.mutable_data();
        }
    }

    Shape shape_wx(sequence, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_wx);

    Shape shape_wh(1, batch_size, 3, hidden_size);
    _temp_WH.try_expand_size(shape_wh);

    anakin_NV_gemm(_cublas_handle, false, false, sequence * batch_size, 3 * hidden_size,
                   _word_size, 1.0, x_data, _weights_i2h.data(), 0.0, _temp_WX.mutable_data());




    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        CUDA_CHECK(hipMemsetAsync(dout_data, 0, sizeof(InDataType) * batch_size * hidden_size,
                                   _ctx->get_compute_stream()));
        h = dout_data;
    } else {
        h = inputs[1]->data();
        CHECK_EQ(inputs[1]->valid_size(), batch_size * hidden_size) <<
                "h size should be batch_size * hidden_size";
    }

    for (int seq = 0; seq < sequence; seq++) {
        const InDataType* hidden_in;
        InDataType* hidden_out = dout_data + seq * batch_size * hidden_size;

        if (seq == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + (seq - 1) * batch_size * hidden_size;
        }

        anakin_NV_gemm(_cublas_handle, false, false, batch_size,
                       3 * hidden_size, hidden_size, 1.0, hidden_in,
                       _weights_h2h.data(), 0.0, _temp_WH.mutable_data());

        OpDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;
        OpDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;
        OpDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;

        OpDataType* w_h_r = _temp_WH.mutable_data() + r_offset * hidden_size;
        OpDataType* w_h_z = _temp_WH.mutable_data() + z_offset * hidden_size;
        OpDataType* w_h_o = _temp_WH.mutable_data() + o_offset * hidden_size;

        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;

        if (param.gate_activity == Active_sigmoid
                && param.h_activity == Active_tanh) {
            cal_one_kernel_sigmoid_tanh_modi_cudnn_formula
                    << < batch_size, frame_per_block, 0, _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_h_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
        } else if (param.gate_activity == Active_sigmoid_fluid
                   && param.h_activity == Active_tanh) {
            cal_one_kernel_paddlesigmoid_tanh_cudnn_formula
                    << < batch_size, frame_per_block, 0, _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_h_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
        } else {
            LOG(ERROR) << "not support active  function";
        }

    }

    if (isHW2Seq) {
        seq2hw(outputs, inputs, param, hidden_size, dout_data);
        outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    }
    return SaberSuccess;

}

template<>
        SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
const std::vector<DataTensor_in*>& inputs,
std::vector<DataTensor_out*>& outputs,
GruParam <OpTensor>& param) {
    if (param.formula == GRU_CUDNN) {
                LOG(ERROR) << "saber cudnn formula not support reverse yet";
        if (param.is_reverse) {
                    LOG(ERROR) << "saber cudnn formula not support reverse yet";

        }
        return gru_cudnn(inputs, outputs, param);
    }

    //    LOG(INFO)<<"gru_paddle";
    DataTensor_in* x = inputs[0];
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* x_data = x->data();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1; //x->get_seq_offset().size()-1;
    int seq_sum = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;

    std::vector<int> emit_offset_vec;
    int emit_length=0;
    _temp_map_dev.try_expand_size(seq_sum);
    isHW2Seq=_seq_util.get_sorted_map(offset,emit_offset_vec,emit_length,_ctx->get_compute_stream());
    if (isHW2Seq) {
        Shape seq_shape(1, 1, seq_sum, _word_size);
        _temp_tensor_in.try_expand_size(seq_shape);
        Shape seq_out_shape(1, 1, seq_sum, _hidden_size);
        _temp_tensor_out.try_expand_size(seq_out_shape);
        _seq_util.seq_2_sorted_seq(x_data,_temp_tensor_in.mutable_data(),_word_size,_ctx->get_compute_stream());
        x_data=_temp_tensor_in.data();
        dout_data = _temp_tensor_out.mutable_data();
    }

    Shape shape_WX(seq_sum, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_WX);

    Shape shape_WH(1, batch_size, 2, hidden_size);
    _temp_WH.try_expand_size(shape_WH);

    Shape shape_WHR(1, batch_size, 1, hidden_size);
    _temp_WHR.try_expand_size(shape_WHR);

    _gemm_wx(seq_sum * batch_size, 3 * hidden_size, _word_size,1.0, x_data,0.0, _weights_i2h.data(),_temp_WX.mutable_data(),_ctx->get_compute_stream());

    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        if(_temp_zero.valid_size()<batch_size * hidden_size){
            _temp_zero.try_expand_size(batch_size * hidden_size);
            CUDA_CHECK(hipMemsetAsync(_temp_zero.mutable_data(), 0, sizeof(OutDataType)*batch_size * hidden_size,
                                       _ctx->get_compute_stream()));
        }

        h = _temp_zero.data();
    } else {
        h = inputs[1]->data();
    }


    for (int word_id = 0; word_id < emit_length; word_id++) {
        int real_word_id = word_id;
        int last_word_id = word_id - 1;

        if (param.is_reverse && batch_size == 1) {
            real_word_id = emit_length - word_id - 1;
            last_word_id = real_word_id + 1;
        }
        int emit_word_id_start = emit_offset_vec[real_word_id];
        int emit_word_id_end = emit_offset_vec[real_word_id + 1];
        int emit_word_length = emit_word_id_end - emit_word_id_start;

        const OutDataType* hidden_in;
        OutDataType* hidden_out = dout_data + emit_offset_vec[real_word_id] * hidden_size;

        if (word_id == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + emit_offset_vec[last_word_id] * hidden_size;
        }

        _gemm_wh_2(emit_word_length, 2 * hidden_size, hidden_size,1.0, hidden_in,0.0, _weights_h2h.data() + hidden_size * hidden_size,_temp_WH.mutable_data(),_ctx->get_compute_stream());

        OutDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;
        OutDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;
        OutDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;

        OutDataType* w_h_r = _temp_WH.mutable_data() + 0 * hidden_size;
        OutDataType* w_h_z = _temp_WH.mutable_data() + 1 * hidden_size;



        const OpDataType * w_o = _weights_h2h.data();
                CHECK_LE(hidden_size, 1024) << "now not support hidden size > 1024 for paddle formula";
        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;
        if(param.gate_activity == Active_sigmoid) {
            RESET_KERNEL_NAME(sigmoid) << < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_r, w_h_r
                            , b_r, hidden_size, hidden_out, hidden_in);
        }else if(param.gate_activity == Active_sigmoid_fluid){
            RESET_KERNEL_NAME(sigmoid_fluid) << < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_r, w_h_r
                            , b_r, hidden_size, hidden_out, hidden_in);
        }else{
            CHECK_EQ(0,1) << "not support gate active  function "<<param.gate_activity;
        }

        _gemm_wh_o(emit_word_length, hidden_size, hidden_size,1.0, hidden_out,0.0,w_o,_temp_WHR.mutable_data(),_ctx->get_compute_stream());

        if(param.gate_activity == Active_sigmoid_fluid&&param.h_activity == Active_tanh_fluid) {
            FINAL_KERNEL_NAME(sigmoid_fluid,tanh_fluid)<< < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_z, w_x_o, w_h_z, b_z, b_o, hidden_size, hidden_out, hidden_in, _temp_WHR.data());
        }else if(param.gate_activity == Active_sigmoid_fluid&&param.h_activity == Active_relu){
            FINAL_KERNEL_NAME(sigmoid_fluid,relu)<< < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_z, w_x_o, w_h_z, b_z, b_o, hidden_size, hidden_out, hidden_in, _temp_WHR.data());
        }else{
            CHECK_EQ(0,1) << "not support active  function "<<param.gate_activity<<","<<param.h_activity;
        }

//        if (param.gate_activity == Active_sigmoid
//            && param.h_activity == Active_tanh) {
//            cal_one_kernel_sigmoid_tanh_paddle_formula
//                    <<< emit_word_length, frame_per_block, sizeof(OutDataType)*hidden_size
//                    , _ctx.get_compute_stream()>>>(
//                    w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
//                            , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
//
//        } else if (param.gate_activity == Active_sigmoid_fluid
//                   && param.h_activity == Active_tanh_fluid) {
//            cal_one_kernel_sigmoidfluid_tanhfluid_paddle_formula
//                    <<< emit_word_length, frame_per_block, sizeof(OutDataType)*hidden_size
//                    , _ctx.get_compute_stream()>>>(
//                    w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
//                            , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
//
//        }  else if (param.gate_activity == Active_sigmoid_fluid
//                    && param.h_activity == Active_relu) {
//            cal_one_kernel_paddlesigmoid_relu_paddle_formula
//                    << < emit_word_length, frame_per_block, sizeof(OutDataType)*hidden_size
//                    , _ctx.get_compute_stream() >> >
//                      (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
//                              , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
//
//        } else {
//                    LOG(ERROR) << "not support active  function";
//        }
    }

    if (isHW2Seq) {
        _seq_util.sorted_seq_2_seq(_temp_tensor_out.data(),dout->mutable_data(),_hidden_size,_ctx->get_compute_stream());
//        LOG(INFO)<<"are you ok";
//        seq2hw(outputs, inputs, param, hidden_size, dout_data);
    }
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}

#if 0
template<>
SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
        const std::vector<DataTensor_in*>& inputs,
        std::vector<DataTensor_out*>& outputs,
        GruParam <OpTensor>& param) {
    if (param.formula == GRU_CUDNN) {
        LOG(ERROR) << "saber cudnn formula not support reverse yet";
        if (param.is_reverse) {
            LOG(ERROR) << "saber cudnn formula not support reverse yet";

        }
        return gru_cudnn(inputs, outputs, param);
    }

    //    LOG(INFO)<<"gru_paddle";
    DataTensor_in* x = inputs[0];
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* x_data = x->data();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1; //x->get_seq_offset().size()-1;
    int sequence = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;

//    CHECK_EQ(w_h2h->height(), hidden_size) << "w_h2h->height()==batch_size";
//    CHECK_EQ(w_h2h->width(), hidden_size * 3) << "w_h2h->width()==hidden_size*3";
//
//    CHECK_EQ(w_i2h->height(), word_size) << "w_i2h->height()==word_size";
//    CHECK_EQ(w_i2h->width(), hidden_size * 3) << "w_i2h->width()==hidden_size*3";

    if (isHW2Seq) {
        x_data = hw2seq(inputs, param, _word_size, hidden_size, sequence);
//        batch_size = inputs[0]->get_seq_offset().size() - 1;

        if (x_data != x->data()) {
            dout_data = _temp_tensor_out.mutable_data();
        }
    }

    Shape shape_WX(sequence, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_WX);

    Shape shape_WH(1, batch_size, 2, hidden_size);
    _temp_WH.try_expand_size(shape_WH);

//    anakin_NV_gemm(_cublas_handle, false, false, sequence * batch_size, 3 * hidden_size,
//                   _word_size, 1.0, x_data, _weights_i2h.data(), 0.0, _temp_WX.mutable_data());

    _gemm_wx(sequence * batch_size, 3 * hidden_size, _word_size,1.0, x_data,0.0, _weights_i2h.data(),_temp_WX.mutable_data(),_ctx.get_compute_stream());

    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        CUDA_CHECK(hipMemsetAsync(dout_data, 0, sizeof(OutDataType)*batch_size * hidden_size,
                                   _ctx->get_compute_stream()));
        h = dout_data;
    } else {
        h = inputs[1]->data();
    }

    for (int seq = 0; seq < sequence; ++seq) {
        int realseq = seq;
        int last_seq = realseq - 1;

        if (param.is_reverse) {
//            DLOG(INFO)<<"reverse gru";
            realseq = sequence - 1 - seq;
            last_seq = realseq + 1;
        }

        const OutDataType* hidden_in;
        OutDataType* hidden_out = dout_data + realseq * batch_size * hidden_size;

        if (seq == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + last_seq * batch_size * hidden_size;
        }

//        anakin_NV_gemm(_cublas_handle, false, false, batch_size,
//                       2 * hidden_size, hidden_size, 1.0, hidden_in,
//                       _weights_h2h.data() + hidden_size * hidden_size, 0.0, _temp_WH.mutable_data());
        _gemm_wh_2(batch_size, 2 * hidden_size, hidden_size,1.0, hidden_in,0.0, _weights_h2h.data() + hidden_size * hidden_size,_temp_WH.mutable_data(),_ctx.get_compute_stream());

        OutDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;
        OutDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;
        OutDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;

        OutDataType* w_h_r = _temp_WH.mutable_data() + 0 * hidden_size;
        OutDataType* w_h_z = _temp_WH.mutable_data() + 1 * hidden_size;
        const OpDataType * w_o = _weights_h2h.data();

        CHECK_LE(hidden_size, 1024) << "now not support hidden size > 1024 for paddle formula";

        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;

        //        DLOG(INFO) << "act = " << param._gate_activity << "," << param._h_activity;

        if (param.gate_activity == Active_sigmoid
                && param.h_activity == Active_tanh) {
            cal_one_kernel_sigmoid_tanh_paddle_formula
            <<< batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
            , _ctx->get_compute_stream()>>>(
                w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        } else if (param.gate_activity == Active_sigmoid_fluid
                   && param.h_activity == Active_tanh_fluid) {
            cal_one_kernel_sigmoidfluid_tanhfluid_paddle_formula
                    <<< batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
                    , _ctx.get_compute_stream()>>>(
                    w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                            , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        }  else if (param.gate_activity == Active_sigmoid_fluid
                    && param.h_activity == Active_relu) {
            cal_one_kernel_paddlesigmoid_relu_paddle_formula
                    << < batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
                    , _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        } else {
            LOG(ERROR) << "not support active  function";
        }
    }

    if (isHW2Seq) {
        seq2hw(outputs, inputs, param, hidden_size, dout_data);
    }
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}
#endif

}
}

