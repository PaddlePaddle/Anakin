#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_gru.h"
#include "saber/core/tensor_op.h"
#include "cuda_inline_activation.h"
namespace anakin {

namespace saber {


static void anakin_NV_gemm(hipblasHandle_t handle, const bool TransA,
                           const bool TransB, const int M, const int N, const int K,
                           const float alpha, const float* A, const float* B, const float beta,
                           float* C) {
    // Note that cublas follows fortran order.
    int lda = (!TransA/* == CblasNoTrans*/) ? K : M;
    int ldb = (!TransB/* == CblasNoTrans*/) ? N : K;
    hipblasOperation_t cuTransA =
        (!TransA/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB =
        (!TransB/* == CblasNoTrans*/) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA,
                             N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <typename Dtype>
__global__ void cal_one_kernel_paddlesigmoid_tanh_cudnn_formula(Dtype* w_x_r, Dtype* w_x_z,
        Dtype* w_x_o,
        Dtype* w_h_r, Dtype* w_h_z, Dtype* w_h_o,
        const Dtype* b_r, const Dtype* b_z, const Dtype* b_o,
        int hidden_size, Dtype* output, const Dtype* hidden_pre) {
    int w_base_index = blockIdx.x * hidden_size * 3;
    int h_base_index = blockIdx.x * hidden_size;
    Dtype* in_w_x_r = w_x_r + w_base_index;
    Dtype* in_w_h_r = w_h_r + w_base_index;
    Dtype* in_w_x_z = w_x_z + w_base_index;
    Dtype* in_w_h_z = w_h_z + w_base_index;
    Dtype* in_w_x_o = w_x_o + w_base_index;
    Dtype* in_w_h_o = w_h_o + w_base_index;
    const Dtype* in_hidden_pre = hidden_pre + h_base_index;
    Dtype* out_output = output + h_base_index;

    for (int index = threadIdx.x; index < hidden_size; index += blockDim.x) {
        const Dtype min = SIGMOID_THRESHOLD_MIN_PADDLE;
        const Dtype max = SIGMOID_THRESHOLD_MAX_PADDLE;

        Dtype before_act_r = in_w_x_r[index] + in_w_h_r[index] + b_r[index];
        before_act_r = (before_act_r < min) ? min : ((before_act_r > max) ? max : before_act_r);
        Dtype act_r = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + exp(-before_act_r));

        Dtype before_act_z = in_w_x_z[index] + in_w_h_z[index] + b_z[index];
        before_act_z = (before_act_z < min) ? min : ((before_act_z > max) ? max : before_act_z);
        Dtype act_z = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + exp(-before_act_z));

        Dtype before_act_h = in_w_x_o[index] + in_w_h_o[index] * act_r
                             + b_o[index];
        before_act_h = (before_act_h > EXP_MAX_INPUT_PADDLE) ? EXP_MAX_INPUT_PADDLE : before_act_h;
        Dtype acted = tanhf(before_act_h);
        out_output[index] = (1 - act_z) * acted + act_z * in_hidden_pre[index];
    }
}

template <typename Dtype>
__global__ void cal_one_kernel_sigmoid_tanh_modi_cudnn_formula(Dtype* w_x_r, Dtype* w_x_z,
        Dtype* w_x_o,
        Dtype* w_h_r, Dtype* w_h_z, Dtype* w_h_o,
        const Dtype* b_r, const Dtype* b_z, const Dtype* b_o,
        int hidden_size, Dtype* output, const Dtype* hidden_pre) {

    int w_base_index = blockIdx.x * hidden_size * 3 + threadIdx.x;
    int h_base_index = blockIdx.x * hidden_size + threadIdx.x;

    for (int index = threadIdx.x; index < hidden_size;
            index += blockDim.x, w_base_index += blockDim.x, h_base_index += blockDim.x) {
        Dtype before_act_r = w_x_r[w_base_index] + w_h_r[w_base_index] + b_r[index];
        Dtype act_r = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-before_act_r));
        Dtype before_act_z = w_x_z[w_base_index] + w_h_z[w_base_index] + b_z[index];
        Dtype act_z = static_cast<Dtype>(1.0) / (static_cast<Dtype>(1.0) + expf(-before_act_z));
        Dtype before_act_h = w_x_o[w_base_index] + w_h_o[w_base_index] * act_r
                             + b_o[index];
        Dtype acted = tanh(before_act_h);
        output[h_base_index] = (static_cast<Dtype>(1.0) - act_z) * acted + act_z * hidden_pre[h_base_index];
    }
}


#define CAL_KERNEL_DEFINE(GATACTNAME)\
template <typename Dtype>\
__global__ void cal_reset_kernel##GATACTNAME(Dtype* w_x_r,Dtype* w_h_r,const Dtype* b_r,int hidden_size, Dtype* output, const Dtype* hidden_pre) {\
    int index = threadIdx.x;\
    if (index > hidden_size) {\
        return;\
    }\
    int w_base_index = blockIdx.x * hidden_size * 3 + index;\
    int u_base_index = blockIdx.x * hidden_size * 2 + index;\
    int h_base_index = blockIdx.x * hidden_size + index;\
    Dtype hidden_pre_value = hidden_pre[h_base_index];\
    Dtype before_act_r = w_x_r[w_base_index] + w_h_r[u_base_index] + b_r[index];\
    Dtype act_r = GATACTNAME(before_act_r);\
    output[h_base_index] = hidden_pre_value * act_r;\
};


#define FINAL_KERNEL_DEFINE(GATACTNAME,OUTACTNAME)\
template <typename Dtype>\
__global__ void cal_final_kernel##GATACTNAME##OUTACTNAME( Dtype* w_x_z, Dtype* w_x_o,Dtype* w_h_z,const Dtype* b_z, const Dtype* b_o,\
        int hidden_size, Dtype* output, const Dtype* hidden_pre,const Dtype* w_h_o) {\
    int index = threadIdx.x;\
    if (index > hidden_size) {\
        return;\
    }\
\
    int w_base_index = blockIdx.x * hidden_size * 3 + index;\
    int u_base_index = blockIdx.x * hidden_size * 2 + index;\
    int h_base_index = blockIdx.x * hidden_size + index;\
    Dtype hidden_pre_value = hidden_pre[h_base_index];\
    Dtype before_act_z = w_x_z[w_base_index] + w_h_z[u_base_index] + b_z[index];\
    Dtype act_z =  GATACTNAME(before_act_z);\
    Dtype before_act_h = w_x_o[w_base_index] + w_h_o[h_base_index]\
                         + b_o[index];\
    Dtype acted = OUTACTNAME(before_act_h);\
\
    output[h_base_index] = (static_cast<Dtype>(1.0) - act_z) * hidden_pre_value + act_z * acted;\
}

#define RESET_KERNEL_NAME(GATACTNAME) cal_reset_kernel##GATACTNAME
#define FINAL_KERNEL_NAME(GATACTNAME,OUTACTNAME) cal_final_kernel##GATACTNAME##OUTACTNAME

CAL_KERNEL_DEFINE(sigmoid);

CAL_KERNEL_DEFINE(sigmoid_fluid);

FINAL_KERNEL_DEFINE(sigmoid_fluid,tanh_fluid);

FINAL_KERNEL_DEFINE(sigmoid_fluid,relu);

#if 0
template <>
SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::gru_cudnn(
    const std::vector<DataTensor_in*> inputs,
    std::vector<DataTensor_out*> outputs,
    GruParam<OpTensor>& param) {

    DataTensor_in* x = inputs[0];
    const InDataType* x_data = x->data();
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1;; //x->get_seq_offset().size()-1;
    int sequence = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;


    if (isHW2Seq) {
        x_data = hw2seq(inputs, param, _word_size, hidden_size, sequence);
        batch_size = offset.size() - 1;

        if (x_data != x->data()) {
            dout_data = _temp_tensor_out.mutable_data();
        }
    }

    Shape shape_wx(sequence, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_wx);

    Shape shape_wh(1, batch_size, 3, hidden_size);
    _temp_WH.try_expand_size(shape_wh);

    anakin_NV_gemm(_cublas_handle, false, false, sequence * batch_size, 3 * hidden_size,
                   _word_size, 1.0, x_data, _weights_i2h.data(), 0.0, _temp_WX.mutable_data());




    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        CUDA_CHECK(hipMemsetAsync(dout_data, 0, sizeof(InDataType) * batch_size * hidden_size,
                                   _ctx->get_compute_stream()));
        h = dout_data;
    } else {
        h = inputs[1]->data();
        CHECK_EQ(inputs[1]->valid_size(), batch_size * hidden_size) <<
                "h size should be batch_size * hidden_size";
    }

    for (int seq = 0; seq < sequence; seq++) {
        const InDataType* hidden_in;
        InDataType* hidden_out = dout_data + seq * batch_size * hidden_size;

        if (seq == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + (seq - 1) * batch_size * hidden_size;
        }

        anakin_NV_gemm(_cublas_handle, false, false, batch_size,
                       3 * hidden_size, hidden_size, 1.0, hidden_in,
                       _weights_h2h.data(), 0.0, _temp_WH.mutable_data());

        OpDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;
        OpDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;
        OpDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                            + seq * batch_size * hidden_size * 3;

        OpDataType* w_h_r = _temp_WH.mutable_data() + r_offset * hidden_size;
        OpDataType* w_h_z = _temp_WH.mutable_data() + z_offset * hidden_size;
        OpDataType* w_h_o = _temp_WH.mutable_data() + o_offset * hidden_size;

        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;

        if (param.gate_activity == Active_sigmoid
                && param.h_activity == Active_tanh) {
            cal_one_kernel_sigmoid_tanh_modi_cudnn_formula
                    << < batch_size, frame_per_block, 0, _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_h_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
        } else if (param.gate_activity == Active_sigmoid_fluid
                   && param.h_activity == Active_tanh) {
            cal_one_kernel_paddlesigmoid_tanh_cudnn_formula
                    << < batch_size, frame_per_block, 0, _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_h_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);
        } else {
            LOG(ERROR) << "not support active  function";
        }

    }

    if (isHW2Seq) {
        seq2hw(outputs, inputs, param, hidden_size, dout_data);
        outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    }
    return SaberSuccess;

}
#endif

template<>
SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
const std::vector<DataTensor_in*>& inputs,
std::vector<DataTensor_out*>& outputs,
GruParam <OpTensor>& param) {
    if (param.formula == GRU_CUDNN) {
//                LOG(ERROR) << "saber cudnn formula not support reverse yet";
//        if (param.is_reverse) {
//                    LOG(ERROR) << "saber cudnn formula not support reverse yet";
//
//        }
//        return gru_cudnn(inputs, outputs, param);
        CHECK(false)<<"not support gru_cudnn now!";
    }

    //    LOG(INFO)<<"gru_paddle";
    DataTensor_in* x = inputs[0];
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* x_data = x->data();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1; //x->get_seq_offset().size()-1;
    int seq_sum = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;

    _temp_map_dev.try_expand_size(seq_sum);
    isHW2Seq = _seq_util.get_sorted_map(offset, _ctx->get_compute_stream());
    auto emit_offset_vec = _seq_util.get_emit_offset_vec();
    auto emit_length = emit_offset_vec.size() - 1;
    if (isHW2Seq) {
        Shape seq_shape(1, 1, seq_sum, _word_size);
        _temp_tensor_in.try_expand_size(seq_shape);
        Shape seq_out_shape(1, 1, seq_sum, _hidden_size);
        _temp_tensor_out.try_expand_size(seq_out_shape);
        _seq_util.seq_2_sorted_seq(x_data,_temp_tensor_in.mutable_data(),_word_size,_ctx->get_compute_stream());
        x_data=_temp_tensor_in.data();
        dout_data = _temp_tensor_out.mutable_data();
    }

    Shape shape_WX(seq_sum, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_WX);

    Shape shape_WH(1, batch_size, 2, hidden_size);
    _temp_WH.try_expand_size(shape_WH);

    Shape shape_WHR(1, batch_size, 1, hidden_size);
    _temp_WHR.try_expand_size(shape_WHR);

    _gemm_wx(seq_sum, 3 * hidden_size, _word_size,1.0, x_data,0.0, _weights_i2h.data(),_temp_WX.mutable_data(),_ctx->get_compute_stream());

    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        if(_temp_zero.valid_size()<batch_size * hidden_size){
            _temp_zero.try_expand_size(batch_size * hidden_size);
            CUDA_CHECK(hipMemsetAsync(_temp_zero.mutable_data(), 0, sizeof(OutDataType)*batch_size * hidden_size,
                                       _ctx->get_compute_stream()));
        }

        h = _temp_zero.data();
    } else {
        h = inputs[1]->data();
    }


    for (int word_id = 0; word_id < emit_length; word_id++) {
        int real_word_id = word_id;
        int last_word_id = word_id - 1;

        if (param.is_reverse && batch_size == 1) {
            real_word_id = emit_length - word_id - 1;
            last_word_id = real_word_id + 1;
        }
        int emit_word_id_start = emit_offset_vec[real_word_id];
        int emit_word_id_end = emit_offset_vec[real_word_id + 1];
        int emit_word_length = emit_word_id_end - emit_word_id_start;

        const OutDataType* hidden_in;
        OutDataType* hidden_out = dout_data + emit_offset_vec[real_word_id] * hidden_size;

        if (word_id == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + emit_offset_vec[last_word_id] * hidden_size;
        }

        _gemm_wh_2(emit_word_length, 2 * hidden_size, hidden_size,1.0, hidden_in,0.0, _weights_h2h.data() + hidden_size * hidden_size,_temp_WH.mutable_data(),_ctx->get_compute_stream());

        OutDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;
        OutDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;
        OutDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                             + emit_word_id_start * hidden_size * 3;

        OutDataType* w_h_r = _temp_WH.mutable_data() + 0 * hidden_size;
        OutDataType* w_h_z = _temp_WH.mutable_data() + 1 * hidden_size;



        const OpDataType * w_o = _weights_h2h.data();
                CHECK_LE(hidden_size, 1024) << "now not support hidden size > 1024 for paddle formula";
        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;
        if(param.gate_activity == Active_sigmoid) {
            RESET_KERNEL_NAME(sigmoid) << < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_r, w_h_r
                            , b_r, hidden_size, hidden_out, hidden_in);
        }else if(param.gate_activity == Active_sigmoid_fluid){
            RESET_KERNEL_NAME(sigmoid_fluid) << < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_r, w_h_r
                            , b_r, hidden_size, hidden_out, hidden_in);
        }else{
            CHECK_EQ(0,1) << "not support gate active  function "<<param.gate_activity;
        }

        _gemm_wh_o(emit_word_length, hidden_size, hidden_size,1.0, hidden_out,0.0,w_o,_temp_WHR.mutable_data(),_ctx->get_compute_stream());

        if(param.gate_activity == Active_sigmoid_fluid&&param.h_activity == Active_tanh_fluid) {
            FINAL_KERNEL_NAME(sigmoid_fluid,tanh_fluid)<< < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_z, w_x_o, w_h_z, b_z, b_o, hidden_size, hidden_out, hidden_in, _temp_WHR.data());
        }else if(param.gate_activity == Active_sigmoid_fluid&&param.h_activity == Active_relu){
            FINAL_KERNEL_NAME(sigmoid_fluid,relu)<< < emit_word_length, frame_per_block, 0
                    , _ctx->get_compute_stream() >> > (
                    w_x_z, w_x_o, w_h_z, b_z, b_o, hidden_size, hidden_out, hidden_in, _temp_WHR.data());
        }else{
            CHECK_EQ(0,1) << "not support active  function "<<param.gate_activity<<","<<param.h_activity;
        }

    }

    if (isHW2Seq) {
        _seq_util.sorted_seq_2_seq(_temp_tensor_out.data(),dout->mutable_data(),_hidden_size,_ctx->get_compute_stream());
    }
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}
template class SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>;
#if 0
template<>
SaberStatus SaberGru<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, NCHW, NCHW, NCHW>::dispatch(\
        const std::vector<DataTensor_in*>& inputs,
        std::vector<DataTensor_out*>& outputs,
        GruParam <OpTensor>& param) {
    if (param.formula == GRU_CUDNN) {
        LOG(ERROR) << "saber cudnn formula not support reverse yet";
        if (param.is_reverse) {
            LOG(ERROR) << "saber cudnn formula not support reverse yet";

        }
        return gru_cudnn(inputs, outputs, param);
    }

    //    LOG(INFO)<<"gru_paddle";
    DataTensor_in* x = inputs[0];
    std::vector<int> offset=x->get_seq_offset();
    const InDataType* x_data = x->data();
    const InDataType* h;
    DataTensor_out* dout = outputs[0];
    OutDataType* dout_data = dout->mutable_data();

    //TODO:check shape first
    const OpTensor* b = param.bias();

    int batch_size = offset.size() - 1; //x->get_seq_offset().size()-1;
    int sequence = x->num();
    int hidden_size = b->valid_size() / 3;
    bool isHW2Seq=offset.size()>2;
    int o_offset = 0;
    int r_offset = 1;
    int z_offset = 2;

//    CHECK_EQ(w_h2h->height(), hidden_size) << "w_h2h->height()==batch_size";
//    CHECK_EQ(w_h2h->width(), hidden_size * 3) << "w_h2h->width()==hidden_size*3";
//
//    CHECK_EQ(w_i2h->height(), word_size) << "w_i2h->height()==word_size";
//    CHECK_EQ(w_i2h->width(), hidden_size * 3) << "w_i2h->width()==hidden_size*3";

    if (isHW2Seq) {
        x_data = hw2seq(inputs, param, _word_size, hidden_size, sequence);
//        batch_size = inputs[0]->get_seq_offset().size() - 1;

        if (x_data != x->data()) {
            dout_data = _temp_tensor_out.mutable_data();
        }
    }

    Shape shape_WX(sequence, batch_size, 3, hidden_size);
    _temp_WX.try_expand_size(shape_WX);

    Shape shape_WH(1, batch_size, 2, hidden_size);
    _temp_WH.try_expand_size(shape_WH);

//    anakin_NV_gemm(_cublas_handle, false, false, sequence * batch_size, 3 * hidden_size,
//                   _word_size, 1.0, x_data, _weights_i2h.data(), 0.0, _temp_WX.mutable_data());

    _gemm_wx(sequence * batch_size, 3 * hidden_size, _word_size,1.0, x_data,0.0, _weights_i2h.data(),_temp_WX.mutable_data(),_ctx.get_compute_stream());

    const OpDataType* b_r = b->data() + r_offset * hidden_size;
    const OpDataType* b_z = b->data() + z_offset * hidden_size;
    const OpDataType* b_o = b->data() + o_offset * hidden_size;

    if (inputs.size() == 1) {
        CUDA_CHECK(hipMemsetAsync(dout_data, 0, sizeof(OutDataType)*batch_size * hidden_size,
                                   _ctx->get_compute_stream()));
        h = dout_data;
    } else {
        h = inputs[1]->data();
    }

    for (int seq = 0; seq < sequence; ++seq) {
        int realseq = seq;
        int last_seq = realseq - 1;

        if (param.is_reverse) {
//            DLOG(INFO)<<"reverse gru";
            realseq = sequence - 1 - seq;
            last_seq = realseq + 1;
        }

        const OutDataType* hidden_in;
        OutDataType* hidden_out = dout_data + realseq * batch_size * hidden_size;

        if (seq == 0) {
            hidden_in = h;
        } else {
            hidden_in = dout_data + last_seq * batch_size * hidden_size;
        }

//        anakin_NV_gemm(_cublas_handle, false, false, batch_size,
//                       2 * hidden_size, hidden_size, 1.0, hidden_in,
//                       _weights_h2h.data() + hidden_size * hidden_size, 0.0, _temp_WH.mutable_data());
        _gemm_wh_2(batch_size, 2 * hidden_size, hidden_size,1.0, hidden_in,0.0, _weights_h2h.data() + hidden_size * hidden_size,_temp_WH.mutable_data(),_ctx.get_compute_stream());

        OutDataType* w_x_r = _temp_WX.mutable_data() + r_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;
        OutDataType* w_x_z = _temp_WX.mutable_data() + z_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;
        OutDataType* w_x_o = _temp_WX.mutable_data() + o_offset * hidden_size
                             + realseq * batch_size * hidden_size * 3;

        OutDataType* w_h_r = _temp_WH.mutable_data() + 0 * hidden_size;
        OutDataType* w_h_z = _temp_WH.mutable_data() + 1 * hidden_size;
        const OpDataType * w_o = _weights_h2h.data();

        CHECK_LE(hidden_size, 1024) << "now not support hidden size > 1024 for paddle formula";

        int frame_per_block = hidden_size <= 1024 ? hidden_size : 1024;

        //        DLOG(INFO) << "act = " << param._gate_activity << "," << param._h_activity;

        if (param.gate_activity == Active_sigmoid
                && param.h_activity == Active_tanh) {
            cal_one_kernel_sigmoid_tanh_paddle_formula
            <<< batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
            , _ctx->get_compute_stream()>>>(
                w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        } else if (param.gate_activity == Active_sigmoid_fluid
                   && param.h_activity == Active_tanh_fluid) {
            cal_one_kernel_sigmoidfluid_tanhfluid_paddle_formula
                    <<< batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
                    , _ctx.get_compute_stream()>>>(
                    w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                            , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        }  else if (param.gate_activity == Active_sigmoid_fluid
                    && param.h_activity == Active_relu) {
            cal_one_kernel_paddlesigmoid_relu_paddle_formula
                    << < batch_size, frame_per_block, sizeof(OutDataType)*hidden_size
                    , _ctx->get_compute_stream() >> >
                    (w_x_r, w_x_z, w_x_o, w_h_r, w_h_z, w_o
                     , b_r, b_z, b_o, hidden_size, hidden_out, hidden_in);

        } else {
            LOG(ERROR) << "not support active  function";
        }
    }

    if (isHW2Seq) {
        seq2hw(outputs, inputs, param, hidden_size, dout_data);
    }
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}
#endif

}
}

